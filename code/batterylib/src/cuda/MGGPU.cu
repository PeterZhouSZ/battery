#include "hip/hip_runtime.h"
#include "MGGPU.cuh"
#include <stdio.h>

#define MAX_CONST_KERNEL_DIM 7
__device__ __constant__ double const_kernel[MAX_CONST_KERNEL_DIM * MAX_CONST_KERNEL_DIM * MAX_CONST_KERNEL_DIM];
__device__ __constant__ int const_kernel_dim;

__device__ __constant__ MGGPU_SysParams const_sys_params;

bool commitSysParams(const MGGPU_SysParams & sysparams) {
	hipError_t res = hipMemcpyToSymbol(HIP_SYMBOL(
		const_sys_params), 
		&sysparams, 
		sizeof(MGGPU_SysParams),
		0,
		hipMemcpyHostToDevice
	);
	return res == hipSuccess;
}

__global__ void ___generateDomain(
	const MGGPU_Volume binaryMask,
	double value_zero,
	double value_one,
	MGGPU_Volume output
) {
	VOLUME_VOX_GUARD(output.res);	

	//Read mask
	uchar c = read<uchar>(binaryMask.surf, vox);		

	//Write value
	write<double>(output.surf, vox, (c > 0) ? value_one : value_zero);
}



void MGGPU_GenerateDomain(
	const MGGPU_Volume & binaryMask,
	double value_zero,
	double value_one,
	MGGPU_Volume & output
) {

	BLOCKS3D(2, output.res);	
	___generateDomain<< < numBlocks, block >> > (
		binaryMask,
		value_zero, 
		value_one, 
		output
	);
}


//Kernel for 3D convolution by 2^3 kernel with stride 2
__global__ void ___convolve3D_2_2(
	const MGGPU_Volume in,	
	const MGGPU_Volume out
){
	//todo shared mem
	VOLUME_VOX_GUARD(out.res);

	double sum = 0.0;

	MGGPU_Kernel3D<2> & k = *((MGGPU_Kernel3D<2>*)const_kernel);
	
	//TODO: boundary handling (upper only)
	sum += read<double>(in.surf, 2 * vox + make_uint3(0, 0, 0)) * k.v[0][0][0];	
	sum += read<double>(in.surf, 2 * vox + make_uint3(0, 0, 1)) * k.v[0][0][1];
	sum += read<double>(in.surf, 2 * vox + make_uint3(0, 1, 0)) * k.v[0][1][0];
	sum += read<double>(in.surf, 2 * vox + make_uint3(0, 1, 1)) * k.v[0][1][1];
	sum += read<double>(in.surf, 2 * vox + make_uint3(1, 0, 0)) * k.v[1][0][0];
	sum += read<double>(in.surf, 2 * vox + make_uint3(1, 0, 1)) * k.v[1][0][1];
	sum += read<double>(in.surf, 2 * vox + make_uint3(1, 1, 0)) * k.v[1][1][0];
	sum += read<double>(in.surf, 2 * vox + make_uint3(1, 1, 1)) * k.v[1][1][1];

	write<double>(out.surf, vox, sum);

}

void MGGPU_Convolve(
	const MGGPU_Volume & in,
	MGGPU_KernelPtr kernel, int kn,
	const MGGPU_Volume & out
) {

	hipError_t res0 = hipMemcpyToSymbol(HIP_SYMBOL(const_kernel), kernel, sizeof(double) * kn * kn * kn, 0, hipMemcpyHostToDevice);
	hipError_t res1 = hipMemcpyToSymbol(HIP_SYMBOL(const_kernel_dim), &kn, sizeof(int), 0, hipMemcpyHostToDevice);
	

	
	BLOCKS3D(2, out.res);
	if (kn == 2) {
		
		___convolve3D_2_2<<< numBlocks, block >>> (
			in,			
			out
			);
	}

}


/*
	Kernel generation
*/

// Lin.sys at top level
template <typename T>
__device__ void MGGPU_GetSystemTopKernel(
	const MGGPU_Volume & domain,
	const uint3 & vox,
	MGGPU_SystemTopKernel * out,
	T * f = nullptr
) {
	
	T Di = read<T>(domain.surf, vox);

	T Dneg[3] = {
		(read<T>(domain.surf, clampedVox(domain.res, vox, X_NEG)) + Di) * T(0.5),
		(read<T>(domain.surf, clampedVox(domain.res, vox, Y_NEG)) + Di) * T(0.5),
		(read<T>(domain.surf, clampedVox(domain.res, vox, Z_NEG)) + Di) * T(0.5)
	};
	T Dpos[3] = {
		(read<T>(domain.surf, clampedVox(domain.res, vox, X_POS)) + Di) * T(0.5),
		(read<T>(domain.surf, clampedVox(domain.res, vox, Y_POS)) + Di) * T(0.5),
		(read<T>(domain.surf, clampedVox(domain.res, vox, Z_POS)) + Di) * T(0.5)
	};
	

	T coeffs[7];	
	bool useInMatrix[7];

	coeffs[DIR_NONE] = T(0);
	useInMatrix[DIR_NONE] = true;

	for (uint j = 0; j < DIR_NONE; j++) {
		const uint k = _getDirIndex(Dir(j));
		const int sgn = _getDirSgn(Dir(j));
		const T Dface = (sgn == -1) ? Dneg[k] : Dpos[k];

		T cellDist[3] = { const_sys_params.cellDim[0],const_sys_params.cellDim[1],const_sys_params.cellDim[2] };
		useInMatrix[j] = true;

		if ((_at<uint>(vox, k) == 0 && sgn == -1) ||
			(_at<uint>(vox, k) == _at<uint>(domain.res, k) - 1 && sgn == 1)
			) {
			cellDist[k] = const_sys_params.cellDim[k] * T(0.5);
			useInMatrix[j] = false;
		}

		coeffs[j] = (Dface * const_sys_params.faceArea[k]) / cellDist[k];

		//Subtract from diagonal
		if (useInMatrix[j] || k == const_sys_params.dirPrimary)
			coeffs[DIR_NONE] -= coeffs[j];
	}


	if (f != nullptr) {
		const uint primaryRes = ((uint*)&domain.res)[const_sys_params.dirPrimary];
		T rhs = T(0);
		if (_at<uint>(vox, const_sys_params.dirPrimary) == 0) {
			Dir dir = _getDir(const_sys_params.dirPrimary, -1);
			rhs -= coeffs[dir] * const_sys_params.concetrationBegin;
		}
		else if (_at<uint>(vox, const_sys_params.dirPrimary) == primaryRes - 1) {
			Dir dir = _getDir(const_sys_params.dirPrimary, 1);
			rhs -= coeffs[dir] * const_sys_params.concetrationEnd;
		}

		*f = rhs;
	}

	#pragma unroll
	for (uint j = 0; j < DIR_NONE; j++) {
		if (!useInMatrix[j])
			coeffs[j] = T(0);
	}
	
	#pragma unroll
	for (uint i = 0; i < 7; i++) {
		out->v[i] = coeffs[i];
	}

	

	
}



__device__  MGGPU_InterpKernel MGGPU_GetInterpolationKernel(
	const MGGPU_Volume & domainSrc,
	const int3 & vox, //vox in destination
	const uint3 & destRes, //should be exactly double (if power of 2) of domain.res
	int dirIndex
) {

	/*
	Two spaces:
	source : n/2 (domain, domain.res)
	dest: n (vox, destRes)
	*/

	MGGPU_InterpKernel kernel;
	
	memset(&kernel, 0, sizeof(MGGPU_InterpKernel));

	//If outside, return zero kernel
	if (!_isValidPos(destRes, vox)) {		
		return kernel;
	}

	

	const int3 r = make_int3(vox.x % 2, vox.y % 2, vox.z % 2) * 2 - 1;	
	const int3 voxSrc = make_int3(vox.x / 2, vox.y / 2, vox.z / 2);

#ifdef DEBUG
	if (!_isValidPos(domainSrc.res, voxSrc)) {
		printf("%d %d %d\n", voxSrc.x, voxSrc.y, voxSrc.z);
	}
#endif


	//Different offset for each subcell
	const int3 offsets[8] = {
		make_int3(0,0,0),
		make_int3(r.x,0,0),
		make_int3(0,r.y,0),
		make_int3(r.x,r.y,0),
		make_int3(0,0,r.z),
		make_int3(r.x,0,r.z),
		make_int3(0,r.y,r.z),
		make_int3(r.x,r.y,r.z)
	};

	//Initial weights
	double P[8] = {
		27, 9, 9, 3, 9, 3, 3, 1
	};

	if ((dirIndex != 0 && (vox.x == destRes.x - 1 || vox.x == 0))) {
		P[0] += P[1]; P[1] = 0;
		P[2] += P[3]; P[3] = 0;
		P[4] += P[5]; P[5] = 0;
		P[6] += P[7]; P[7] = 0;
	}

	if ((dirIndex != 1 && (vox.y == destRes.y - 1 || vox.y == 0))) {
		P[0] += P[2]; P[2] = 0;
		P[1] += P[3]; P[3] = 0;
		P[4] += P[6]; P[6] = 0;
		P[5] += P[7]; P[7] = 0;
	}

	if ((dirIndex != 2 && (vox.z == destRes.z - 1 || vox.z == 0))) {
		P[0] += P[4]; P[4] = 0;
		P[1] += P[5]; P[5] = 0;
		P[2] += P[6]; P[6] = 0;
		P[3] += P[7]; P[7] = 0;
	}


	

	double w[8];
	double W = 0.0;
	for (int i = 0; i < 8; i++) {
		if (P[i] == 0) continue;
		w[i] = P[i];


		int3 voxSrcNew = voxSrc + offsets[i];
		if (_isValidPos(domainSrc.res, voxSrcNew)) {
			w[i] *= read<double>(domainSrc.surf, make_uint3(voxSrcNew)); //redundant conversion to uint, TODO better
		}
		//Source voxel is outside of domain
		//P[i] > 0 then implies it's on dirichlet boundary
		//Therefore a nearest value has to be used
		else {

			//Change offset to nearest valid voxel from source
			int3 offset = offsets[i];
			
			_at<int, int3>(offset, dirIndex) += 1;
			if (!_isValidPos(domainSrc.res, voxSrc + offset)) {
				_at<int, int3>(offset, (dirIndex + 1) % 3) += 1;
			}

			if (!_isValidPos(domainSrc.res, voxSrc + offset)) {
				_at<int, int3>(offset, (dirIndex + 2) % 3) += 1;
			}

			//Update src vox with new offset
			voxSrcNew = voxSrc + offset;

			if(!_isValidPos(domainSrc.res, voxSrc + offset)) {
				int3 p = voxSrc + offset;
				printf("%d %d %d\n", p.x, p.y, p.z);
			}

			//Read weight from source domain
			//w[i] *= read<double>(domainSrc.surf, make_uint3(voxSrcNew));
		}

		W += w[i];
	}


	

	//Normalize weights
	for (auto i = 0; i < 8; i++) {
		w[i] /= W;
	}

	

	//Create 3^3 kernel
	memset(kernel.v, 0, INTERP_SIZE*INTERP_SIZE*INTERP_SIZE * sizeof(double));
	for (auto i = 0; i < 8; i++) {
		int3 kpos = make_int3(1, 1, 1) + offsets[i];
		kernel.v[kpos.x][kpos.y][kpos.z] = w[i];
	}



	return kernel;
}





void __global__ ___systemTopKernel(
	MGGPU_Volume domain,
	MGGPU_SystemTopKernel * A0,
	MGGPU_Volume f
){
	VOLUME_VOX_GUARD(domain.res);

	size_t i = _linearIndex(domain.res, vox);

	double fval = 0.0;
	MGGPU_GetSystemTopKernel<double>(domain, vox, &A0[i], &fval);	
	write<double>(f.surf, vox, fval);

}

void MGGPU_GenerateSystemTopKernel(
	const MGGPU_Volume & domain,
	MGGPU_SystemTopKernel * A0,
	MGGPU_Volume & f
) {

	BLOCKS3D(2, domain.res);
	___systemTopKernel << < numBlocks, block >> > (
		domain,
		A0,
		f
		);
	


}







void __device__ MGGPU_Convolve_A0_I_Direct(
	const MGGPU_Volume & domain,
	const MGGPU_SystemTopKernel * A0,
	const uint3 & vox,
	int dirIndex,
	MGGPU_Kernel3D<5> * out
) {
	const int3 voxi = make_int3(vox);

	const int N_A = 3;
	const int N_I = 3;
	const int N_AI = N_A + N_I - 1; //5

	MGGPU_Kernel3D<N_AI> & AI = *out;


	//Read packed a0 kernel
	size_t i = _linearIndex(domain.res, vox);
	const MGGPU_SystemTopKernel & a7 = A0[i];

	//Scatter to 3x3x3 kernel
	MGGPU_Kernel3D<3> a;
	{
		memset(&a, 0, sizeof(MGGPU_Kernel3D<3>));
		a.v[1][1][1] = a7.v[DIR_NONE];
		a.v[0][1][1] = a7.v[X_NEG];
		a.v[2][1][1] = a7.v[X_POS];
		a.v[1][0][1] = a7.v[Y_NEG];
		a.v[1][2][1] = a7.v[Y_POS];
		a.v[1][1][0] = a7.v[Z_NEG];
		a.v[1][1][2] = a7.v[Z_POS];
	}



	for (int x_ai = 0; x_ai < N_AI; x_ai++) {
		for (int y_ai = 0; y_ai < N_AI; y_ai++) {
			for (int z_ai = 0; z_ai < N_AI; z_ai++) {
				int3 offsetACenter = make_int3(-N_AI / 2) + make_int3(x_ai, y_ai, z_ai);


				//Get I kernel at _ai pos
				MGGPU_InterpKernel I = MGGPU_GetInterpolationKernel(domain, voxi + offsetACenter, domain.res, dirIndex);

				double sum = 0.0;

				//dot with offseted a
				for (int x_i = 0; x_i < N_I; x_i++) {
					for (int y_i = 0; y_i < N_I; y_i++) {
						for (int z_i = 0; z_i < N_I; z_i++) {

							int3 offsetICenter = make_int3(-N_I / 2) + make_int3(x_i, y_i, z_i);

							int3 apos = make_int3(N_A / 2) + offsetACenter + offsetICenter;
							int x_a = apos.x;
							int y_a = apos.y;
							int z_a = apos.z;

							if (!_isValidPos(make_uint3(N_A), make_int3(x_a, y_a, z_a)))
								continue;

							sum += I.v[x_i][y_i][z_i] * a.v[x_a][y_a][z_a];
						}
					}
				}

				AI.v[x_ai][y_ai][z_ai] = sum;
			}
		}
	}


}



__global__ void ___convolve_A0_I_Direct(
	const MGGPU_Volume domain,
	const MGGPU_SystemTopKernel * A0,
	MGGPU_Kernel3D<5> * AI
	){

	VOLUME_VOX_GUARD(domain.res);

	size_t i = _linearIndex(domain.res, vox);

	MGGPU_Convolve_A0_I_Direct(domain, A0, vox, const_sys_params.dirPrimary, &(AI[i]) );


}

void MGGPU_GenerateAI0(
	const MGGPU_Volume & domain,
	const MGGPU_SystemTopKernel * A0,
	MGGPU_Kernel3D<5> * AI
) {


	BLOCKS3D(2, domain.res);
	___convolve_A0_I_Direct << < numBlocks, block >> > (
		domain,
		A0,
		AI
		);

}


__global__ void ___genI(
	const uint3 destRes,
	const MGGPU_Volume domainHalf,	
	MGGPU_InterpKernel * I
) {

	VOLUME_VOX_GUARD(destRes);	
	size_t i = _linearIndex(destRes, vox);

	const int3 ivox = make_int3(vox);
	I[i] = MGGPU_GetInterpolationKernel(domainHalf, ivox, destRes, const_sys_params.dirPrimary);

}


void MGGPU_GenerateSystemInterpKernels(
	const uint3 & destRes, 
	const MGGPU_Volume & domainHalf,
	MGGPU_InterpKernel * I
) {
	BLOCKS3D(2, destRes);
	___genI << < numBlocks, block >> > (
		destRes,
		domainHalf,
		I
		);
}