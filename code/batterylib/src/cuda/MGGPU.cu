#include "hip/hip_runtime.h"
#include "MGGPU.cuh"
#include <stdio.h>

#define MAX_CONST_KERNEL_DIM 7

struct KernelCombineParams {
	uint3 resArow;
	uint3 resAcol;
	uint3 resBrow;
	uint3 resBcol;
	uint3 resCrow;
	uint3 resCcol;
	MGGPU_KernelPtr A,B,C;	
	int Adim, Bdim, Cdim;	
	int Aratio, Bratio, Cratio;
	int AdimHalf, BdimHalf, CdimHalf;

	MGGPU_Volume domain;
};


__device__ __constant__ double const_kernel[MAX_CONST_KERNEL_DIM * MAX_CONST_KERNEL_DIM * MAX_CONST_KERNEL_DIM];
__device__ __constant__ int const_kernel_dim;

__device__ __constant__ MGGPU_SysParams const_sys_params;

__device__ __constant__ KernelCombineParams const_kernel_combine_params;




bool commitSysParams(const MGGPU_SysParams & sysparams) {
	hipError_t res = hipMemcpyToSymbol(HIP_SYMBOL(
		const_sys_params), 
		&sysparams, 
		sizeof(MGGPU_SysParams),
		0,
		hipMemcpyHostToDevice
	);
	return res == hipSuccess;
}

bool commitKernelCombineParams(const KernelCombineParams & p) {
	hipError_t res = hipMemcpyToSymbol(HIP_SYMBOL(
		const_kernel_combine_params),
		&p,
		sizeof(KernelCombineParams),
		0,
		hipMemcpyHostToDevice
	);
	if (res != hipSuccess) {
		printf("Failed to commit kernel combine params\n");
		exit(1);
	}
	return res == hipSuccess;
}

__global__ void ___generateDomain(
	const MGGPU_Volume binaryMask,
	double value_zero,
	double value_one,
	MGGPU_Volume output
) {
	VOLUME_VOX_GUARD(output.res);	

	//Read mask
	uchar c = read<uchar>(binaryMask.surf, vox);		

	//Write value
	write<double>(output.surf, vox, (c > 0) ? value_one : value_zero);
}



void MGGPU_GenerateDomain(
	const MGGPU_Volume & binaryMask,
	double value_zero,
	double value_one,
	MGGPU_Volume & output
) {

	BLOCKS3D(2, output.res);	
	___generateDomain<< < numBlocks, block >> > (
		binaryMask,
		value_zero, 
		value_one, 
		output
	);
}


//Kernel for 3D convolution by 2^3 kernel with stride 2 and border
__global__ void ___convolve3D_2_2(
	const MGGPU_Volume in,	
	const MGGPU_Volume out
){
	//todo shared mem
	VOLUME_VOX_GUARD(out.res);

	double sum = 0.0;

	MGGPU_Kernel3D<2> & k = *((MGGPU_Kernel3D<2>*)const_kernel);
	
	//TODO: boundary handling (upper only)
	int3 s = make_int3(1, 1, 1); //stride
	const uint3 voxSrc = vox * 2;
	if (voxSrc.x == in.res.x - 1)
		s.x = 0;
	if (voxSrc.y == in.res.y - 1)
		s.y = 0;
	if (voxSrc.z == in.res.z - 1)
		s.z = 0;

	sum += read<double>(in.surf, voxSrc + make_uint3(0, 0, 0)) * k.v[0][0][0];
	sum += read<double>(in.surf, voxSrc + make_uint3(0, 0, s.z)) * k.v[0][0][1];
	sum += read<double>(in.surf, voxSrc + make_uint3(0, s.y, 0)) * k.v[0][1][0];
	sum += read<double>(in.surf, voxSrc + make_uint3(0, s.y, s.z)) * k.v[0][1][1];
	sum += read<double>(in.surf, voxSrc + make_uint3(s.x, 0, 0)) * k.v[1][0][0];
	sum += read<double>(in.surf, voxSrc + make_uint3(s.x, 0, s.z)) * k.v[1][0][1];
	sum += read<double>(in.surf, voxSrc + make_uint3(s.x, s.y, 0)) * k.v[1][1][0];
	sum += read<double>(in.surf, voxSrc + make_uint3(s.x, s.y, s.z)) * k.v[1][1][1];

	write<double>(out.surf, vox, sum);

}

void MGGPU_Convolve(
	const MGGPU_Volume & in,
	MGGPU_KernelPtr kernel, int kn,
	const MGGPU_Volume & out
) {

	hipError_t res0 = hipMemcpyToSymbol(HIP_SYMBOL(const_kernel), kernel, sizeof(double) * kn * kn * kn, 0, hipMemcpyHostToDevice);
	hipError_t res1 = hipMemcpyToSymbol(HIP_SYMBOL(const_kernel_dim), &kn, sizeof(int), 0, hipMemcpyHostToDevice);
	

	
	BLOCKS3D(2, out.res);
	if (kn == 2) {
		
		___convolve3D_2_2<<< numBlocks, block >>> (
			in,			
			out
			);
	}

}


/*
	Kernel generation
*/

// Lin.sys at top level
template <typename T>
__device__ void MGGPU_GetSystemTopKernel(
	const MGGPU_Volume & domain,
	const uint3 & vox,
	MGGPU_SystemTopKernel * out,
	T * f = nullptr
) {
	
	T Di = read<T>(domain.surf, vox);

	T Dneg[3] = {
		(read<T>(domain.surf, clampedVox(domain.res, vox, X_NEG)) + Di) * T(0.5),
		(read<T>(domain.surf, clampedVox(domain.res, vox, Y_NEG)) + Di) * T(0.5),
		(read<T>(domain.surf, clampedVox(domain.res, vox, Z_NEG)) + Di) * T(0.5)
	};
	T Dpos[3] = {
		(read<T>(domain.surf, clampedVox(domain.res, vox, X_POS)) + Di) * T(0.5),
		(read<T>(domain.surf, clampedVox(domain.res, vox, Y_POS)) + Di) * T(0.5),
		(read<T>(domain.surf, clampedVox(domain.res, vox, Z_POS)) + Di) * T(0.5)
	};
	

	T coeffs[7];	
	bool useInMatrix[7];

	coeffs[DIR_NONE] = T(0);
	useInMatrix[DIR_NONE] = true;

	for (uint j = 0; j < DIR_NONE; j++) {
		const uint k = _getDirIndex(Dir(j));
		const int sgn = _getDirSgn(Dir(j));
		const T Dface = (sgn == -1) ? Dneg[k] : Dpos[k];

		T cellDist[3] = { const_sys_params.cellDim[0],const_sys_params.cellDim[1],const_sys_params.cellDim[2] };
		useInMatrix[j] = true;

		if ((_at<uint>(vox, k) == 0 && sgn == -1) ||
			(_at<uint>(vox, k) == _at<uint>(domain.res, k) - 1 && sgn == 1)
			) {
			cellDist[k] = const_sys_params.cellDim[k] * T(0.5);
			useInMatrix[j] = false;
		}

		coeffs[j] = (Dface * const_sys_params.faceArea[k]) / cellDist[k];

		//Subtract from diagonal
		if (useInMatrix[j] || k == const_sys_params.dirPrimary)
			coeffs[DIR_NONE] -= coeffs[j];
	}


	if (f != nullptr) {
		const uint primaryRes = ((uint*)&domain.res)[const_sys_params.dirPrimary];
		T rhs = T(0);
		if (_at<uint>(vox, const_sys_params.dirPrimary) == 0) {
			Dir dir = _getDir(const_sys_params.dirPrimary, -1);
			rhs -= coeffs[dir] * const_sys_params.concetrationBegin;
		}
		else if (_at<uint>(vox, const_sys_params.dirPrimary) == primaryRes - 1) {
			Dir dir = _getDir(const_sys_params.dirPrimary, 1);
			rhs -= coeffs[dir] * const_sys_params.concetrationEnd;
		}

		*f = rhs;
	}

	#pragma unroll
	for (uint j = 0; j < DIR_NONE; j++) {
		if (!useInMatrix[j])
			coeffs[j] = T(0);
	}
	
	#pragma unroll
	for (uint i = 0; i < 7; i++) {
		out->v[i] = coeffs[i];
	}

	

	
}





__device__ __host__  MGGPU_InterpKernel MGGPU_GetInterpolationKernel(
	const MGGPU_Volume & domainSrc,
	const int3 & vox, //vox in destination
	const uint3 & destRes, //should be exactly double (if power of 2) of domain.res
	int dirIndex
) {

	MGGPU_InterpKernel kernel;

	/*
	Two spaces:
	source : n/2 (domain, domain.res)
	dest: n (vox, destRes)
	*/

	
	
	memset(&kernel, 0, sizeof(MGGPU_InterpKernel));

	//If outside, return zero kernel
	if (!_isValidPos(destRes, vox)) {		
		return kernel;
	}

	

	const int3 r = make_int3(vox.x % 2, vox.y % 2, vox.z % 2) * 2 - 1;	
	const int3 voxSrc = make_int3(vox.x / 2, vox.y / 2, vox.z / 2);

#ifdef DEBUG
	if (!_isValidPos(domainSrc.res, voxSrc)) {
		printf("%d %d %d\n", voxSrc.x, voxSrc.y, voxSrc.z);
	}
#endif


	//Different offset for each subcell
	const int3 offsets[8] = {
		make_int3(0,0,0),
		make_int3(r.x,0,0),
		make_int3(0,r.y,0),
		make_int3(r.x,r.y,0),
		make_int3(0,0,r.z),
		make_int3(r.x,0,r.z),
		make_int3(0,r.y,r.z),
		make_int3(r.x,r.y,r.z)
	};

	//Initial weights
	double P[8] = {
		27, 9, 9, 3, 9, 3, 3, 1
	};

	if ((dirIndex != 0 && (vox.x == destRes.x - 1 || vox.x == 0))) {
		P[0] += P[1]; P[1] = 0;
		P[2] += P[3]; P[3] = 0;
		P[4] += P[5]; P[5] = 0;
		P[6] += P[7]; P[7] = 0;
	}

	if ((dirIndex != 1 && (vox.y == destRes.y - 1 || vox.y == 0))) {
		P[0] += P[2]; P[2] = 0;
		P[1] += P[3]; P[3] = 0;
		P[4] += P[6]; P[6] = 0;
		P[5] += P[7]; P[7] = 0;
	}

	if ((dirIndex != 2 && (vox.z == destRes.z - 1 || vox.z == 0))) {
		P[0] += P[4]; P[4] = 0;
		P[1] += P[5]; P[5] = 0;
		P[2] += P[6]; P[6] = 0;
		P[3] += P[7]; P[7] = 0;
	}


	

	double w[8] = {
		0,0,0,0,0,0,0,0
	};
	double W = 0.0;
	for (int i = 0; i < 8; i++) {
		if (P[i] == 0) continue;
		w[i] = P[i];


		int3 voxSrcNew = voxSrc + offsets[i];
		if (_isValidPos(domainSrc.res, voxSrcNew)) {
#ifdef __CUDA_ARCH__
			w[i] *= read<double>(domainSrc.surf, make_uint3(voxSrcNew)); //redundant conversion to uint, TODO better
#else
			w[i] *= ((double*)domainSrc.cpu)[_linearIndex(domainSrc.res, voxSrcNew)];
#endif
		}
		//Source voxel is outside of domain
		//P[i] > 0 then implies it's on dirichlet boundary
		//Therefore a nearest value has to be used
		else {

			//Change offset to nearest valid voxel from source
			int3 offset = offsets[i];
			
			_at<int, int3>(offset, dirIndex) = 0;
			if (!_isValidPos(domainSrc.res, voxSrc + offset)) {
				_at<int, int3>(offset, (dirIndex + 1) % 3) = 0;
			}

			if (!_isValidPos(domainSrc.res, voxSrc + offset)) {
				_at<int, int3>(offset, (dirIndex + 2) % 3) = 0;
			}

			//Update src vox with new offset
			voxSrcNew = voxSrc + offset;
#ifdef DEBUG
			if(!_isValidPos(domainSrc.res, voxSrc + offset)) {
				int3 p = voxSrc + offset;
				printf("%d %d %d\n", p.x, p.y, p.z);
			}
#endif

			//Read weight from source domain
#ifdef __CUDA_ARCH__
			w[i] *= read<double>(domainSrc.surf, make_uint3(voxSrcNew));
#else
			w[i] *= ((double*)domainSrc.cpu)[_linearIndex(domainSrc.res, voxSrcNew)];
#endif
		}

		W += w[i];
	}


	

	//Normalize weights
	for (auto i = 0; i < 8; i++) {
		w[i] /= W;
	}

	

	//Create 3^3 kernel
	memset(kernel.v, 0, INTERP_SIZE*INTERP_SIZE*INTERP_SIZE * sizeof(double));
	for (auto i = 0; i < 8; i++) {
		int3 kpos = make_int3(1, 1, 1) + offsets[i];
		kernel.v[kpos.x][kpos.y][kpos.z] = w[i];
	}


	return kernel;
}





void __global__ ___systemTopKernel(
	MGGPU_Volume domain,
	MGGPU_SystemTopKernel * A0,
	MGGPU_Volume f
){
	VOLUME_VOX_GUARD(domain.res);

	size_t i = _linearIndex(domain.res, vox);

	double fval = 0.0;
	MGGPU_GetSystemTopKernel<double>(domain, vox, &A0[i], &fval);	
	write<double>(f.surf, vox, fval);

}

void MGGPU_GenerateSystemTopKernel(
	const MGGPU_Volume & domain,
	MGGPU_SystemTopKernel * A0,
	MGGPU_Volume & f
) {

	BLOCKS3D(2, domain.res);
	___systemTopKernel << < numBlocks, block >> > (
		domain,
		A0,
		f
		);
	


}







void __device__ MGGPU_Convolve_A0_I_Direct(
	const uint3 destRes,
	const MGGPU_Volume & domain,
	const MGGPU_SystemTopKernel * A0,
	const uint3 & vox,
	int dirIndex,
	MGGPU_Kernel3D<5> * out
) {
	const int3 voxi = make_int3(vox);

	const int N_A = 3;
	const int N_I = 3;
	const int N_AI = N_A + N_I - 1; //5

	MGGPU_Kernel3D<N_AI> & AI = *out;


	//Read packed a0 kernel
	size_t i = _linearIndex(destRes, vox);
	const MGGPU_SystemTopKernel & a7 = A0[i];

	//Scatter to 3x3x3 kernel
	MGGPU_Kernel3D<3> a;
	{
		memset(&a, 0, sizeof(MGGPU_Kernel3D<3>));
		a.v[1][1][1] = a7.v[DIR_NONE];
		a.v[0][1][1] = a7.v[X_NEG];
		a.v[2][1][1] = a7.v[X_POS];
		a.v[1][0][1] = a7.v[Y_NEG];
		a.v[1][2][1] = a7.v[Y_POS];
		a.v[1][1][0] = a7.v[Z_NEG];
		a.v[1][1][2] = a7.v[Z_POS];
	}



	for (int x_ai = 0; x_ai < N_AI; x_ai++) {
		for (int y_ai = 0; y_ai < N_AI; y_ai++) {
			for (int z_ai = 0; z_ai < N_AI; z_ai++) {
				int3 offsetACenter = make_int3(-N_AI / 2) + make_int3(x_ai, y_ai, z_ai);


				//Get I kernel at _ai pos
				MGGPU_InterpKernel I = MGGPU_GetInterpolationKernel(domain, voxi + offsetACenter, destRes, dirIndex);

				double sum = 0.0;

				//dot with offseted a
				for (int x_i = 0; x_i < N_I; x_i++) {
					for (int y_i = 0; y_i < N_I; y_i++) {
						for (int z_i = 0; z_i < N_I; z_i++) {

							int3 offsetICenter = make_int3(-N_I / 2) + make_int3(x_i, y_i, z_i);

							int3 apos = offsetACenter + make_int3(N_A / 2) + offsetICenter;
							int x_a = apos.x;
							int y_a = apos.y;
							int z_a = apos.z;

							

							if (!_isValidPos(make_uint3(N_A), make_int3(x_a, y_a, z_a)))
								continue;

							sum += I.v[x_i][y_i][z_i] * a.v[x_a][y_a][z_a];
						}
					}
				}

				AI.v[x_ai][y_ai][z_ai] = sum;
			}
		}
	}


}



__global__ void ___convolve_A0_I_Direct(
	const uint3 destRes,
	const MGGPU_Volume domainHalf,
	const MGGPU_SystemTopKernel * A0,
	MGGPU_Kernel3D<5> * AI
	){

	VOLUME_VOX_GUARD(destRes);

	size_t i = _linearIndex(destRes, vox);

	MGGPU_Convolve_A0_I_Direct(destRes, domainHalf, A0, vox, const_sys_params.dirPrimary, &(AI[i]) );


}

void MGGPU_GenerateAI0(
	const MGGPU_Volume & domainHalf,
	const MGGPU_SystemTopKernel * A0,
	MGGPU_Kernel3D<5> * AI
) {

	const uint3 destRes = domainHalf.res * 2; // TODO:

	BLOCKS3D(2, destRes);
	___convolve_A0_I_Direct << < numBlocks, block >> > (
		destRes,
		domainHalf,
		A0,
		AI
		);

}


__global__ void ___genI(
	const uint3 destRes,
	const MGGPU_Volume domainHalf,	
	MGGPU_InterpKernel * I
) {

	VOLUME_VOX_GUARD(destRes);	
	size_t i = _linearIndex(destRes, vox);

	const int3 ivox = make_int3(vox);
	I[i] = MGGPU_GetInterpolationKernel(domainHalf, ivox, destRes, const_sys_params.dirPrimary);

}


void MGGPU_GenerateSystemInterpKernels(
	const uint3 & destRes, 
	const MGGPU_Volume & domainHalf,
	MGGPU_InterpKernel * I
) {
	BLOCKS3D(2, destRes);
	___genI << < numBlocks, block >> > (
		destRes,
		domainHalf,
		I
		);
}



__global__ void ___genITranpose(
	const uint3 Nres,
	const uint3 Nhalfres,
	const MGGPU_Volume domainHalf,
	MGGPU_Kernel3D<4> * output
){

		 
	VOLUME_VOX_GUARD(Nres);	
	int3 ivox = make_int3(vox);
	int3 ivoxhalf = make_int3(vox.x / 2, vox.y / 2, vox.z / 2);

	//Get kernel interpolating to ivox from Nhalfres
	MGGPU_InterpKernel kernel = MGGPU_GetInterpolationKernel(domainHalf, ivox, Nres, const_sys_params.dirPrimary);

	//Scatter kernel
	for (int i = 0; i < 3; i++) {
		for (int j = 0; j < 3; j++) {
			for (int k = 0; k < 3; k++) {
				double val = kernel.v[i][j][k];

				int3 targetHalf = ivoxhalf + make_int3(i - 1, j - 1, k - 1);
				size_t debugLinPosHalf = _linearIndex(Nhalfres, targetHalf);


				if (_isValidPos(Nhalfres, targetHalf)){
					//output index
					size_t index = _linearIndex(Nhalfres, targetHalf);

					//inoutput index
					int3 inOutputIndex = ivox - (targetHalf * 2 - 1);
					
					if (!_isValidPos(make_uint3(4), inOutputIndex)) {
						continue;
					}

					if (index == 0) {
						//printf("(%d, (%d, %d, %d) / (%d %d %d))\t %f\n", index, inOutputIndex.x, inOutputIndex.y, inOutputIndex.z, ivox.x, ivox.y, ivox.z, val);
					}
					output[index].v[inOutputIndex.x][inOutputIndex.y][inOutputIndex.z] = val;
				}
				
			}
		}
	}



}



void MGGPU_GenerateTranposeInterpKernels(
	const uint3 & Nres,
	const uint3 & Nhalfres,
	const MGGPU_Volume & domainHalf,
	MGGPU_Kernel3D<4> * output
) {

	
	BLOCKS3D(2, Nres);
	___genITranpose << < numBlocks, block >> > (
		Nres, Nhalfres,
		domainHalf,
		output
		);

}

__global__ void ___convolve_A0_IT0_Direct(
	const uint3 Nres,
	const uint3 Nhalfres,
	const MGGPU_SystemTopKernel * A0,
	const MGGPU_Kernel3D<4> * IT0,
	MGGPU_Kernel3D<3> * output
){

	VOLUME_VOX_GUARD(Nres);

	size_t iN = _linearIndex(Nres, vox);
	size_t iNHalf = _linearIndex(Nhalfres, vox);


	int3 voxi = make_int3(vox);
	int3 voxiHalf = make_int3(voxi.x /2, voxi.y / 2, voxi.z / 2);

	const int N_A = 3;	
	const int N_I = 4;
	const int STRIDE = 2;
	const int N_AI = (N_A + N_I - 1) / STRIDE; //3	


	//Read packed a0 kernel
	size_t i = _linearIndex(Nres, vox);
	MGGPU_Kernel3D<N_AI> & AI = output[i];

	const MGGPU_SystemTopKernel & a7 = A0[i];

	//Scatter to 3x3x3 kernel
	MGGPU_Kernel3D<3> a;
	{
		memset(&a, 0, sizeof(MGGPU_Kernel3D<3>));
		a.v[1][1][1] = a7.v[DIR_NONE];
		a.v[0][1][1] = a7.v[X_NEG];
		a.v[2][1][1] = a7.v[X_POS];
		a.v[1][0][1] = a7.v[Y_NEG];
		a.v[1][2][1] = a7.v[Y_POS];
		a.v[1][1][0] = a7.v[Z_NEG];
		a.v[1][1][2] = a7.v[Z_POS];
	}
	

	for (int x_ai = 0; x_ai < N_AI; x_ai++) {
		for (int y_ai = 0; y_ai < N_AI; y_ai++) {
			for (int z_ai = 0; z_ai < N_AI; z_ai++) {

				int3 offsetAICenter = make_int3(-N_AI / 2) + make_int3(x_ai, y_ai, z_ai);
				int3 interpPos = voxiHalf + offsetAICenter;
				
				if (!_isValidPos(Nhalfres, interpPos)) {
					AI.v[x_ai][y_ai][z_ai] = 0.0;
					continue;
				}

				size_t interpIndex = _linearIndex(Nhalfres, voxiHalf + offsetAICenter);

				const MGGPU_Kernel3D<N_I> & I = IT0[interpIndex];

				double sum = 0.0;
				//dot with offseted a
				for (int x_i = 0; x_i < N_I; x_i++) {
					for (int y_i = 0; y_i < N_I; y_i++) {
						for (int z_i = 0; z_i < N_I; z_i++) {

						


							int3 offsetICenter = make_int3(x_i, y_i, z_i) - make_int3(N_I / 2 - 1);
							int3 voxI = 2*(interpPos) + offsetICenter;
							
							
							int3 apos = voxI - voxi + make_int3(1,1,1);

							
							int x_a = apos.x;
							int y_a = apos.y;
							int z_a = apos.z;

							if (!_isValidPos(make_uint3(N_A), make_int3(x_a, y_a, z_a)))
								continue;
							
							sum += I.v[x_i][y_i][z_i] * a.v[x_a][y_a][z_a];
						}
					}
				}

				AI.v[x_ai][y_ai][z_ai] = sum;
			}
		}
	}


}



void MGGPU_GenerateAI0(
	const uint3 & Nres,
	const uint3 & Nhalfres,
	const MGGPU_SystemTopKernel * A0,
	const MGGPU_Kernel3D<4> * IT0,
	MGGPU_Kernel3D<3> * output
) {
	
	BLOCKS3D(2, Nres);
	___convolve_A0_IT0_Direct << < numBlocks, block >> > (
		Nres,
		Nhalfres,
		A0,
		IT0,
		output
		);

}


__global__ void __genA1(
	const uint3 Nres,
	const uint3 Nhalfres,
	const MGGPU_Kernel3D<4> * IT0,
	MGGPU_Kernel3D<5> * output
) {
	VOLUME_VOX_GUARD(Nhalfres);


	/*int3 ivox = make_int3(vox);
	//int3 ivoxHalf = make_int3(vox.x / 2, vox.y / 2, vox.z / 2);

	size_t i = _linearIndex(Nhalfres, ivox);

	MGGPU_RestrictKernel R = MGGPU_GetRestrictionKernel(vox, Nhalfres, const_sys_params.dirPrimary);

	for (auto x_r = 0; x_r < RESTR_SIZE; x_r++) {
		for (auto y_r = 0; y_r < RESTR_SIZE; y_r++) {
			for (auto z_r = 0; z_r < RESTR_SIZE; z_r++) {
				
				int3 kVox = 2*ivox + make_int3(x_r, y_r, z_r) - make_int3(RESTR_SIZE / 2 - 1);
				if (!_isValidPos(Nres, kVox))
					continue;
				
				size_t ki = _linearIndex(kVox, Nres);

			}
		}
	}

	*/


}


void MGGPU_GenerateA1(
	const uint3 & Nres,
	const uint3 & Nhalfres,	
	const MGGPU_Kernel3D<4> * IT0,
	MGGPU_Kernel3D<5> * output
) {

	BLOCKS3D(2, Nhalfres);
	__genA1 << < numBlocks, block >> > (
		Nres,
		Nhalfres,		
		IT0,
		output
		);

}



//////////////////////////////////////////////

enum CombineType {
	A_IS_GENERIC,
	A_IS_TOPLEVEL,
	A_IS_RESTRICTION
};


//combineKernelsAt<A_IS_GENERIC,5*5*5>
//combineKernelsAt<A_IS_TOP_LEVEL,7>
//combineKernelsAt<A_IS_RESTRICTION,4*4*4>

__device__ __host__ void combineKernelDotInner() {

}

template<CombineType combineType, size_t AKernelAllocSize = 5*5*5>
__device__ __host__ void combineKernelsAt(
	int3 ivox, 
	const KernelCombineParams & p
) {


	const int Asize = p.Adim*p.Adim*p.Adim;
	const int Bsize= p.Bdim*p.Bdim*p.Bdim;
	const int Csize= p.Cdim*p.Cdim*p.Cdim;
	
	//

	//i in matrix multiply
	size_t Crow = _linearIndex(p.resCrow, ivox); // in resCrow == resArow space
	const size_t Arow = Crow; // in resCrow == resArow space

	
	
	double kernelAStore[AKernelAllocSize];
	if (combineType == A_IS_GENERIC) {
		memcpy(kernelAStore, p.A + Asize * Arow, Asize * sizeof(double));
	}
	else if (combineType == A_IS_TOPLEVEL) {
		memcpy(kernelAStore, ((char*)p.A) + sizeof(MGGPU_SystemTopKernel) * Arow, sizeof(MGGPU_SystemTopKernel));
	}
	else if (combineType == A_IS_RESTRICTION) {		
		MGGPU_GetRestrictionKernel(make_uint3(ivox), p.resArow, const_sys_params.dirPrimary, kernelAStore);
	}

	const MGGPU_KernelPtr kernelA = kernelAStore;
	const MGGPU_KernelPtr kernelC = p.C + Csize * Crow;

	//Columns of C				
	for (int ck = -p.CdimHalf; ck < p.Cdim - p.CdimHalf; ck++) {
		for (int cj = -p.CdimHalf; cj < p.Cdim - p.CdimHalf; cj++) {
			for (int ci = -p.CdimHalf; ci < p.Cdim - p.CdimHalf; ci++) {

				//j in matrix multipy (only nonzero result cols)
				/*Get voxel pos in Crow space
				-> project ivox to Crow space, then apply kernel delta
				*/


				int3 ivoxCcol = make_int3(ivox.x / p.Cratio, ivox.y / p.Cratio, ivox.z / p.Cratio) + make_int3(ci, cj, ck);
				int3 ivoxBcol = ivoxCcol;

				if (!_isValidPos(p.resCcol, ivoxCcol)) {
					continue;
				}

				size_t Bcol = _linearIndex(p.resBcol, ivoxBcol);

				//multiply / dot product
				//Arow=Crow * Bcol=Ccol

				double sum = 0.0;

				if (combineType != A_IS_TOPLEVEL) {					
					for (int ak = -p.AdimHalf; ak < p.Adim - p.AdimHalf; ak++) {
						for (int aj = -p.AdimHalf; aj < p.Adim - p.AdimHalf; aj++) {
							for (int ai = -p.AdimHalf; ai < p.Adim - p.AdimHalf; ai++) {

								int3 ivoxDot = ivox * p.Aratio + make_int3(ai, aj, ak);
								if (!_isValidPos(p.resAcol, ivoxDot)) {
									continue;
								}

								size_t iDot = _linearIndex(p.resBrow, ivoxDot);

								double valA = 0.0;
								valA = kernelA[
									_linearIndexXFirst(make_uint3(p.Adim), make_int3(ai + p.AdimHalf, aj + p.AdimHalf, ak + p.AdimHalf))
								];


								MGGPU_KernelPtr kernelB = p.B + Bsize * iDot;
								int3 BkernelOffset = ivoxBcol - make_int3(ivoxDot.x / p.Bratio, ivoxDot.y / p.Bratio, ivoxDot.z / p.Bratio);
								int bi = BkernelOffset.x + p.BdimHalf;
								int bj = BkernelOffset.y + p.BdimHalf;
								int bk = BkernelOffset.z + p.BdimHalf;

								if (!_isValidPos(make_uint3(p.Bdim), make_int3(bi, bj, bk))) {
									continue;
								}

								double valB = kernelB[
									_linearIndexXFirst(make_uint3(p.Bdim), make_int3(bi, bj, bk))
								];
								sum += valA*valB;

							}
						}
					}
				}
				else {
					for (int dir = 0; dir <= DIR_NONE; dir++) {

						int3 ivoxDot = ivox * p.Aratio + dirVec(Dir(dir)); //make_int3(ai, aj, ak);
						if (!_isValidPos(p.resAcol, ivoxDot)) {
							continue;
						}

						

						

						double valA = kernelA[dir];						

						MGGPU_InterpKernel kernelInterp = MGGPU_GetInterpolationKernel(p.domain, ivoxDot, p.resBrow, const_sys_params.dirPrimary);
						MGGPU_KernelPtr kernelB = MGGPU_KernelPtr(&kernelInterp);

						

						int3 BkernelOffset = ivoxBcol - make_int3(ivoxDot.x / p.Bratio, ivoxDot.y / p.Bratio, ivoxDot.z / p.Bratio);
						int bi = BkernelOffset.x + p.BdimHalf;
						int bj = BkernelOffset.y + p.BdimHalf;
						int bk = BkernelOffset.z + p.BdimHalf;

						if (!_isValidPos(make_uint3(p.Bdim), make_int3(bi, bj, bk))) {
							continue;
						}


						double valB = kernelB[
							_linearIndexXFirst(make_uint3(p.Bdim), make_int3(bi, bj, bk))
						];

						/*{
							size_t iDot = _linearIndex(p.resBrow, ivoxDot);
							MGGPU_KernelPtr kernelB0 = p.B + Bsize * iDot;
							double valB0 = kernelB0[
								_linearIndexXFirst(make_uint3(p.Bdim), make_int3(bi, bj, bk))
							];

							double diff = valB0 - valB;
							if (diff > 0.0) {
								printf("%f\n", diff);
							}
						}*/

						

						sum += valA*valB;

					}
				}

				if (sum != 0.0) {
					kernelC[
						_linearIndexXFirst(make_uint3(p.Cdim), make_int3(ci + p.CdimHalf, cj + p.CdimHalf, ck + p.CdimHalf))
					] = sum;
				}

			}
		}
	}

	


}

template<CombineType combineType, size_t AKernelAllocSize = 5 * 5 * 5>
__global__ void __combineKernels()
{
	//Get voxel position
	VOLUME_VOX_GUARD(const_kernel_combine_params.resArow);
	int3 ivox = make_int3(vox);
	KernelCombineParams & p = const_kernel_combine_params;
	//Combine kernels at ivox position
	combineKernelsAt<combineType, AKernelAllocSize>(ivox, p);		

}


bool MGGPU_CombineKernelsTopLevel(
	const uint3 resA,
	const uint3 resBrow,
	const uint3 resBcol,
	const MGGPU_KernelPtr A,
	const MGGPU_KernelPtr B,
	const int Bdim, 
	MGGPU_KernelPtr C,
	MGGPU_Volume interpDomain,
	bool onDevice
){

	if (resA.x != resBrow.x ||
		resA.y != resBrow.y ||
		resA.z != resBrow.z) {
		return false;
	}

	KernelCombineParams p;
	p.resArow = resA;
	p.resAcol = resA;
	p.resBrow = resBrow;
	p.resBcol = resBcol;

	p.resCrow = resA;
	p.resCcol = resBcol;

	p.A = A;
	p.B = B;
	p.C = C;

	p.Bratio = p.resBrow.x / p.resBcol.x;
	p.Aratio = p.resAcol.x / p.resArow.x;
	p.Cratio = p.resCrow.x / p.resCcol.x;

	p.Adim = 3;
	p.Bdim = Bdim;
	p.Cdim = MGGPU_outputKernelSize(p.Adim, p.Bdim, p.Bratio);

	p.CdimHalf = (p.Cdim % 2 == 0) ? p.Cdim / 2 - 1 : p.Cdim / 2;
	p.BdimHalf = (p.Bdim % 2 == 0) ? p.Bdim / 2 - 1 : p.Bdim / 2;
	p.AdimHalf = (p.Adim % 2 == 0) ? p.Adim / 2 - 1 : p.Adim / 2;
	
	p.domain = interpDomain;

	const CombineType combineType = A_IS_TOPLEVEL;
	const size_t allocA = sizeof(MGGPU_SystemTopKernel) / sizeof(double);

	if (onDevice) {
		commitKernelCombineParams(p);
		BLOCKS3D(2, p.resCrow);
		__combineKernels<combineType, allocA> << < numBlocks, block >> > ();
	}
	else {
		int3 ivox;
		for (ivox.z = 0; ivox.z < p.resCrow.z; ivox.z++) {
			for (ivox.y = 0; ivox.y < p.resCrow.y; ivox.y++) {
				for (ivox.x = 0; ivox.x < p.resCrow.x; ivox.x++) {
					combineKernelsAt<combineType, allocA>(ivox, p);
				}
			}
		}
	}


}


bool MGGPU_CombineKernelsRestrict(
	const uint3 resArow,
	const uint3 resAcol,
	const uint3 resBrow,
	const uint3 resBcol,	
	const MGGPU_KernelPtr B,
	const int Bdim,
	MGGPU_KernelPtr C,
	bool onDevice
) {

	

	if (resAcol.x != resBrow.x ||
		resAcol.y != resBrow.y ||
		resAcol.z != resBrow.z) {
		return false;
	}

	KernelCombineParams p;
	p.resArow = resArow;
	p.resAcol = resAcol;
	p.resBrow = resBrow;
	p.resBcol = resBcol;

	p.resCrow = resArow;
	p.resCcol = resBcol;

	p.A = nullptr;
	p.B = B;
	p.C = C;

	p.Bratio = p.resBrow.x / p.resBcol.x;
	p.Aratio = p.resAcol.x / p.resArow.x;
	p.Cratio = p.resCrow.x / p.resCcol.x;

	p.Adim = RESTR_SIZE;
	p.Bdim = Bdim;	
	p.Cdim = MGGPU_outputKernelSize(p.Adim, p.Bdim, p.Bratio);

	p.CdimHalf = (p.Cdim % 2 == 0) ? p.Cdim / 2 - 1 : p.Cdim / 2;
	p.BdimHalf = (p.Bdim % 2 == 0) ? p.Bdim / 2 - 1 : p.Bdim / 2;
	p.AdimHalf = (p.Adim % 2 == 0) ? p.Adim / 2 - 1 : p.Adim / 2;

	const CombineType combineType = A_IS_RESTRICTION;
	const size_t allocA = RESTR_SIZE*RESTR_SIZE*RESTR_SIZE;


	if (onDevice) {
		commitKernelCombineParams(p);
		BLOCKS3D(4, p.resCrow);
		__combineKernels<combineType, allocA> << < numBlocks, block >> > ();
	}
	else {
		int3 ivox;
		for (ivox.z = 0; ivox.z < p.resCrow.z; ivox.z++) {
			for (ivox.y = 0; ivox.y < p.resCrow.y; ivox.y++) {
				for (ivox.x = 0; ivox.x < p.resCrow.x; ivox.x++) {
					combineKernelsAt<combineType, allocA>(ivox, p);
				}
			}
		}
	}

	return true;


}


bool MGGPU_CombineKernelsGeneric(
	const uint3 resArow,
	const uint3 resAcol,	
	const uint3 resBrow,
	const uint3 resBcol,	
	const MGGPU_KernelPtr A,
	const int Adim,
	const MGGPU_KernelPtr B,
	const int Bdim, // in resBcol
	MGGPU_KernelPtr C,
	bool onDevice
) {
	
	if (resAcol.x != resBrow.x ||
		resAcol.y != resBrow.y ||
		resAcol.z != resBrow.z) {
		return false;
	}

	KernelCombineParams p;
	p.resArow = resArow;
	p.resAcol = resAcol;
	p.resBrow = resBrow;
	p.resBcol = resBcol;

	p.resCrow = resArow;
	p.resCcol = resBcol;

	p.A = A;
	p.B = B;
	p.C = C;
		
	p.Bratio = p.resBrow.x / p.resBcol.x;
	p.Aratio = p.resAcol.x / p.resArow.x;
	p.Cratio = p.resCrow.x / p.resCcol.x;

	p.Adim = Adim;
	p.Bdim = Bdim;
	p.Cdim = MGGPU_outputKernelSize(p.Adim, p.Bdim, p.Bratio);

	p.CdimHalf = (p.Cdim % 2 == 0) ? p.Cdim / 2 - 1 : p.Cdim / 2;
	p.BdimHalf = (p.Bdim % 2 == 0) ? p.Bdim / 2 - 1 : p.Bdim / 2;
	p.AdimHalf = (p.Adim % 2 == 0) ? p.Adim / 2 - 1 : p.Adim / 2;

	const CombineType combineType = A_IS_GENERIC;
	const size_t allocA = 5*5*5;


	if (onDevice) {
		commitKernelCombineParams(p);
		BLOCKS3D(4, p.resCrow);
		__combineKernels<combineType, allocA> << < numBlocks, block >> > ();
	}
	else {
		int3 ivox;
		for (ivox.z = 0; ivox.z < p.resCrow.z; ivox.z++) {
			for (ivox.y = 0; ivox.y < p.resCrow.y; ivox.y++) {
				for (ivox.x = 0; ivox.x < p.resCrow.x; ivox.x++) {
					combineKernelsAt<combineType, allocA>(ivox, p);
				}
			}
		}	
	}
	

	return true;
}
