#include "hip/hip_runtime.h"
#include "VolumeCCL.cuh"
#include <stdio.h>
#include <assert.h>

#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <set>

/*
	Based on
	Ohira, N. (2018). Memory-efficient 3D connected component labeling with parallel computing. Signal, Image and Video Processing, 12(3), 429-436.
*/


struct VolumeCCL_Params {
	size_t spanCount;

	//Input resolution	
	uint3 res;

	//Count of spans for each row in y-z plane
	uint * spanCountPlane;

	//Resolution of span/label volumes
	uint3 spanRes;	
	uint2 * spanVolume;
	uint * labelVolume;
	
	//Buffer for inclusive scan of labels
	uint * labelScan;

	bool * hasChanged;
};


template <typename T>
inline __device__ bool opEqual(const T & a, const T &  b, const T & threshold = 0) {
	return (a == b);
}


template <typename T>
using CmpOp = bool(*)(
	const T & a, const T & b, const T & threshold
	);



template <typename T, CmpOp<T> _cmpOp>
__global__ void __countSpansKernel(
	CUDA_Volume input, 
	uint * result,
	T background,
	T threshold
) {

	const uint3 resultRes = make_uint3(1, input.res.y, input.res.z);
	VOLUME_IVOX_GUARD(resultRes);
	

	uint spanSum = 0;
	int3 curPos = ivox;

	T prevVal = background;

	bool open = false;
	for (int i = 0; i < input.res.x; i++) {
		T newVal = read<T>(input.surf, curPos);
		
		if (!_cmpOp(prevVal, newVal, threshold)) {
			open = !open;
			if (open) {
				spanSum++;				
			}
		}		
		prevVal = newVal;
		curPos.x++;
	}

	size_t linIndex = _linearIndex(resultRes, ivox);
	result[linIndex] = spanSum;

}


/*
	Detect spans again and assign them to span and label matrices
*/
template <typename T, CmpOp<T> _cmpOp>
__global__ void __buildMatricesKernel(
	CUDA_Volume input,
	VolumeCCL_Params p,
	T background,
	T threshold
) {
	const uint3 planeRes = make_uint3(1, input.res.y, input.res.z);
	VOLUME_IVOX_GUARD(planeRes);	

	uint currentSpanIndex = 0;
	int3 curPos = ivox;

	T prevVal = background;
	uint spanBegin = 0;

	bool open = false;
	for (int i = 0; i < input.res.x; i++) {
		T newVal = read<T>(input.surf, curPos);

		if (!_cmpOp(prevVal,newVal,threshold)) {
			if (!open) {				
				spanBegin = i;								
			}
			else {		
				size_t index = _linearIndex(p.spanRes, make_int3(currentSpanIndex, ivox.y, ivox.z));
				p.spanVolume[index] = make_uint2(spanBegin, i - 1);				
				p.labelVolume[index] = _linearIndex(input.res, make_uint3(spanBegin, ivox.y, ivox.z));
				currentSpanIndex++;
			}
			open = !open;
		}
		prevVal = newVal;
		curPos.x++;
	}

	//Close last one if open
	if (open) {
		size_t index = _linearIndex(p.spanRes, make_int3(currentSpanIndex, ivox.y, ivox.z));
		p.spanVolume[index] = make_uint2(spanBegin, input.res.x - 1);
		p.labelVolume[index] = _linearIndex(input.res, make_uint3(spanBegin, ivox.y, ivox.z));
	}

}


inline __device__ bool spanOverlap(uint2 a, uint2 b) {
	return a.x <= b.y && b.x <= a.y;
}

inline __device__ uint labelEquivalence(VolumeCCL_Params & p, uint index, int3 ivox) {

	
	
	uint indexPrev = 0;	
	do {
		indexPrev = index;	

		//Find pos of label
		uint3 posOrig = posFromLinear(p.res, index);
		uint3 posLabel = make_uint3(0, posOrig.y, posOrig.z);

		const uint rowSpanCount = p.spanCountPlane[_linearIndex(make_uint3(1, p.res.y, p.res.z), posLabel)];
		for (int i = 0; i < rowSpanCount; i++) {			
			uint2 span = p.spanVolume[_linearIndex(p.spanRes, posLabel)];
			if (posOrig.x >= span.x && posOrig.x <= span.y)
				break;		
			posLabel.x++;
		}
		index = p.labelVolume[_linearIndex(p.spanRes, posLabel)];
		
	}
	while(index != indexPrev);

	return index;
}

//input surface not needed
__global__ void __updateContinuityKernel(VolumeCCL_Params p) {
	const uint3 planeRes = make_uint3(1, p.res.y, p.res.z);
	VOLUME_IVOX_GUARD(planeRes);

	const uint rowSpanCount = p.spanCountPlane[_linearIndex(planeRes, ivox)];
	
	int3 curPosSpan = ivox;
	

	const int3 offsets[4] = {
		{ 0,-1,0 },
		{ 0,1,0 },
		{ 0,0,-1 },
		{ 0,0,1 },
	};

	int3 otherPosSpan[4] = {
		ivox + offsets[0], ivox + offsets[1], ivox + offsets[2], ivox + offsets[3]
	};



	const uint otherSpanCount[4] = {
		(ivox.y == 0) ?				0 : p.spanCountPlane[_linearIndex(planeRes, ivox + offsets[0])],
		(ivox.y == p.res.y - 1) ?	0 : p.spanCountPlane[_linearIndex(planeRes, ivox + offsets[1])],
		(ivox.z == 0) ?				0 : p.spanCountPlane[_linearIndex(planeRes, ivox + offsets[2])],
		(ivox.z == p.res.z - 1) ?	0 : p.spanCountPlane[_linearIndex(planeRes, ivox + offsets[3])]
	};

	for (int i = 0; i < rowSpanCount; i++) {
		uint2 thisSpan = p.spanVolume[_linearIndex(p.spanRes, curPosSpan)];		
		uint tempLabel = p.labelVolume[_linearIndex(p.spanRes, curPosSpan)];
		

		

		#pragma unroll
		for (int k = 0; k < 4; k++) {
			

			if (k == 0 && ivox.y == 0) continue;
			if (k == 1 && ivox.y == p.res.y - 1) continue;
			if (k == 2 && ivox.z == 0) continue;
			if (k == 3 && ivox.z == p.res.z - 1) continue;
	
			
			while(otherPosSpan[k].x < otherSpanCount[k]){
				uint2 otherSpan = p.spanVolume[_linearIndex(p.spanRes, otherPosSpan[k])];

				

				if (otherSpan.x > thisSpan.y) break;							

				if (spanOverlap(thisSpan, otherSpan)) {										
					uint * thisLabelPtr = p.labelVolume + _linearIndex(p.spanRes, curPosSpan);
					uint * otherLabelPtr = p.labelVolume + _linearIndex(p.spanRes, otherPosSpan[k]);

					uint thisLabel = *thisLabelPtr;
					uint otherLabel = *otherLabelPtr;
					
					if (thisLabel < otherLabel) {
						atomicMin(otherLabelPtr, thisLabel);
						*p.hasChanged = true;
					}
					else if(otherLabel < thisLabel) {
						atomicMin(thisLabelPtr, otherLabel);
						*p.hasChanged = true;
					}				
					
				}

				otherPosSpan[k].x++;				
			}
		}	

		curPosSpan.x++;
		

	}


	curPosSpan = ivox;	
	size_t index0 = _linearIndex(p.res, ivox);	

	for (int i = 0; i < rowSpanCount; i++) {
		uint index = p.labelVolume[_linearIndex(p.spanRes, curPosSpan)];


		p.labelVolume[_linearIndex(p.spanRes, curPosSpan)] = labelEquivalence(p, index, ivox);

		curPosSpan.x++;		
	}


}



__global__ void __labelOutputKernel(
	VolumeCCL_Params p,
	CUDA_Volume output
	
) {
	const uint3 planeRes = make_uint3(1, p.res.y, p.res.z);
	VOLUME_IVOX_GUARD(planeRes);

	const uint rowSpanCount = p.spanCountPlane[_linearIndex(planeRes, ivox)];

	int3 posSpan = ivox;
	for (; posSpan.x < rowSpanCount; posSpan.x++) {

		const uint2 thisSpan = p.spanVolume[_linearIndex(p.spanRes, posSpan)];
		const uint thisLabel = p.labelVolume[_linearIndex(p.spanRes, posSpan)];
		
		for (int k = thisSpan.x; k <= thisSpan.y; k++) {
			const int3 pos = make_int3(k, ivox.y, ivox.z);
			write<uint>(output.surf, pos, thisLabel);
		}		
	}

}

__global__ void __markRootLabelsKernel(VolumeCCL_Params p){	
	const uint3 planeRes = make_uint3(1, p.res.y, p.res.z);
	VOLUME_IVOX_GUARD(planeRes);

	const uint rowSpanCount = p.spanCountPlane[_linearIndex(planeRes, ivox)];

	int3 posSpan = ivox;
	for (; posSpan.x < rowSpanCount; posSpan.x++) {
		const size_t spanIndex = _linearIndex(p.spanRes, posSpan);
		const uint2 thisSpan = p.spanVolume[spanIndex];
		const uint thisLabel = p.labelVolume[spanIndex];

		const uint spanBeginIndex = _linearIndex(p.res, ivox + make_int3(thisSpan.x, 0, 0));		
		p.labelScan[spanIndex] = (spanBeginIndex == thisLabel) ? 1 : 0;		
	}

}

__global__ void __reindexLabels(VolumeCCL_Params p) {
	const uint3 planeRes = make_uint3(1, p.res.y, p.res.z);
	VOLUME_IVOX_GUARD(planeRes);

	const uint rowSpanCount = p.spanCountPlane[_linearIndex(planeRes, ivox)];

	int3 posSpan = ivox;
	for (; posSpan.x < rowSpanCount; posSpan.x++) {
		const size_t spanIndex = _linearIndex(p.spanRes, posSpan);
		//const uint2 thisSpan = p.spanVolume[spanIndex];
		const uint thisLabel = p.labelVolume[spanIndex];
		
		//Find original label row
		uint3 origLabelPos = posFromLinear(p.res, thisLabel);
		const uint origXPos = origLabelPos.x;
		origLabelPos.x = 0;
		
		uint newLabel = uint(-1);
		//Iterate through the row to find original index (based on span begin)
		const uint otherRowSpanCount = p.spanCountPlane[_linearIndex(planeRes, origLabelPos)];
		for (; origLabelPos.x < otherRowSpanCount; origLabelPos.x++){
			size_t otherIndex = _linearIndex(p.spanRes, origLabelPos);
			//When span found, get scanned label (new unique label) 
			if (origXPos == p.spanVolume[otherIndex].x) {
				newLabel = p.labelScan[otherIndex];
			}
		}

		p.labelVolume[spanIndex] = newLabel;		
	}

}

#ifdef _DEBUG 
#define DEBUG_CPU
#endif

uint VolumeCCL(const CUDA_Volume & input, CUDA_Volume & output, uchar background)
{
	assert(input.type == TYPE_UCHAR);
	//assert(output.type == TYPE_UINT);
	
	VolumeCCL_Params p;	
	p.res = input.res;

	uint numLabels = 0;

	thrust::device_vector<uint> sumPlane(input.res.y * input.res.z);
	p.spanCountPlane = sumPlane.data().get();

	//Count spands
	{
		BLOCKS3D_INT3(1, 8, 8, make_uint3(1, input.res.y, input.res.z));
		//Summing in ascending X direction
		
		if (input.type == TYPE_UCHAR) {			
			uchar threshold = 1;
			__countSpansKernel<uchar, opEqual<uchar>> << < numBlocks, block >> > (input, p.spanCountPlane, background, threshold);
		}
		else {
			assert("Unsupported type");
			exit(0);
		}
		uint maxSpanCount = thrust::reduce(sumPlane.begin(), sumPlane.end(), 0, thrust::maximum<uint>());


		p.spanCount = maxSpanCount;
		p.spanRes = make_uint3(p.spanCount, p.res.y, p.res.z);		
		
	}
#ifdef DEBUG_CPU
	thrust::host_vector<uint> hostSum = sumPlane;
	uint * dataSum = hostSum.data();
#endif
	

	{
		

		thrust::device_vector<uint2> spanMatrix(p.spanRes.x * p.spanRes.y * p.spanRes.z);
		thrust::device_vector<uint> labelMatrix(p.spanRes.x * p.spanRes.y * p.spanRes.z);
		thrust::device_vector<uint> labelScan(p.spanRes.x * p.spanRes.y * p.spanRes.z);

		p.spanVolume = spanMatrix.data().get();
		p.labelVolume = labelMatrix.data().get();
		p.labelScan = labelScan.data().get();

#ifdef DEBUG_CPU
		thrust::host_vector<uint2> hostSpan;
		thrust::host_vector<uint> hostLabel;
		thrust::host_vector<uint> hostLabelScan;

		hipDeviceSetLimit(hipLimitPrintfFifoSize, 1024 * 1024 * 64);
#endif

		hipMalloc(&p.hasChanged, 1);
		hipMemset(p.hasChanged, 0, 1);

		{
			BLOCKS3D_INT3(1, 8, 8, make_uint3(1, input.res.y, input.res.z));
			if (input.type == TYPE_UCHAR) {				
				uchar threshold = 1;				
				__buildMatricesKernel<uchar, opEqual<uchar>> << < numBlocks, block >> > (input, p, background, threshold);			
			}
			else {
				assert("Unsupported type");
				exit(0);
			}
			
		}

#ifdef DEBUG_CPU
		{
			hostSpan = spanMatrix;  uint2 * dataSpan = hostSpan.data();
			hostLabel = labelMatrix; uint * dataLabel = hostLabel.data();
			char b;
			b = 0;
		}

		
#endif
		std::set<uint> uniqueLabels;

		bool hasChangedHost = false;
		int iteration = 0;
		do
		{
			BLOCKS3D_INT3(1, 8, 8, make_uint3(1, input.res.y, input.res.z));
			__updateContinuityKernel << < numBlocks, block >> >(p);
			hipMemcpy(&hasChangedHost, p.hasChanged, 1, hipMemcpyDeviceToHost);
			hipMemset(p.hasChanged, 0, 1);

#ifdef DEBUG_CPU
			{
								
				hostSpan = spanMatrix;  uint2 * dataSpan = hostSpan.data();
				hostLabel = labelMatrix; uint * dataLabel = hostLabel.data();
				
				uniqueLabels.clear();
				for (auto i = 0; i < p.spanRes.x * p.spanRes.y * p.spanRes.z; i++) {
					uniqueLabels.insert(hostLabel[i]);
				}				

				
				char b;
				b = 0;

				iteration++;
			}
#endif

		} while (hasChangedHost);		
		hipFree(p.hasChanged);


#ifdef DEBUG_CPU
		bool isZeroALabel = false;
		{
			uint firstCount;
			uint firstLabel;
			uint2 firstSpan;

			hipMemcpy(&firstCount, p.spanCountPlane, sizeof(uint), hipMemcpyDeviceToHost);
			hipMemcpy(&firstLabel, p.labelVolume, sizeof(uint), hipMemcpyDeviceToHost);
			hipMemcpy(&firstSpan, p.spanVolume, sizeof(uint2), hipMemcpyDeviceToHost);

			if (firstCount > 0 && firstLabel == 0 && firstSpan.x == 0) {
				isZeroALabel = true;
			}
		}
		
		if (!isZeroALabel) { 
			uniqueLabels.erase(0);
		}
		
		for (auto & s : uniqueLabels) {
			printf("%u, ", s);
		}
		printf("\n");
#endif

		


		//reindexing
		{
			//label matrix where first labeled (label == data idnex) is marked as 1, otherwise 0
			{
				BLOCKS3D_INT3(1, 8, 8, make_uint3(1, input.res.y, input.res.z));
				__markRootLabelsKernel << < numBlocks, block >> > (p);

#ifdef DEBUG_CPU
				hostLabelScan = labelScan;
				uint * dataLabelScan = hostLabelScan.data();
				char b;
				b = 0;
#endif
			}

			//inclusive scan			
			{
				thrust::inclusive_scan(labelScan.begin(), labelScan.end(), labelScan.begin());

#ifdef DEBUG_CPU
				hostLabelScan = labelScan;
				uint * dataLabelScan = hostLabelScan.data();
				char b;
				b = 0;
#endif
			}
			//get number of unique labels
			
			{
				hipMemcpy(&numLabels, p.labelScan + p.spanRes.x * p.spanRes.y * p.spanRes.z - 1, sizeof(uint), hipMemcpyDeviceToHost);
			}

			printf("Label count: %u\n", numLabels);			
			
			//reindex
			{
				BLOCKS3D_INT3(1, 8, 8, make_uint3(1, input.res.y, input.res.z));
				__reindexLabels << < numBlocks, block >> > (p);

#ifdef DEBUG_CPU
				{

					hostSpan = spanMatrix;  uint2 * dataSpan = hostSpan.data();
					hostLabel = labelMatrix; uint * dataLabel = hostLabel.data();

					uniqueLabels.clear();
					for (auto i = 0; i < p.spanRes.x * p.spanRes.y * p.spanRes.z; i++) {
						uniqueLabels.insert(hostLabel[i]);
					}

					for (auto & s : uniqueLabels) {
						printf("%u, ", s);
					}
					printf("\n");

				}
#endif
			}

		}

		//Reconstruct volume
		/*{
			assert(output.type == TYPE_UCHAR);
			BLOCKS3D_INT3(1, 8, 8, make_uint3(1, input.res.y, input.res.z));
			__labelOutputKernelUchar << < numBlocks, block >> > (p, output, label);
		}*/

		{
			assert(output.type == TYPE_UCHAR);
			BLOCKS3D_INT3(1, 8, 8, make_uint3(1, input.res.y, input.res.z));
			__labelOutputKernel << < numBlocks, block >> > (p, output);
		}

		//Need outputs:
		/*
			a) uint, 0 = background, 1..N labels (for storage/representation)
			b) float3, colormap/randomized -> outside of this function (for visualization)
			c) uchar, 0 none, 1 selected label(s) -> for reactive area density / filtering
		*/
		

	}

	


	return numLabels;
}




