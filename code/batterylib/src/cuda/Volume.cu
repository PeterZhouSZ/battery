#include "hip/hip_runtime.h"
#include "Volume.cuh"
#include <stdio.h>


	



__global__ void kernelErode(uint3 res, hipSurfaceObject_t surfIn, hipSurfaceObject_t surfOut) {
	
	VOLUME_VOX_GUARD(res);

	float vals[6];	
	surf3Dread(&vals[0], surfIn, ((vox.x - 1 + res.x) % res.x) * sizeof(float), vox.y, vox.z);
	surf3Dread(&vals[1], surfIn, ((vox.x + 1) % res.x) * sizeof(float), vox.y, vox.z);
	surf3Dread(&vals[2], surfIn, vox.x * sizeof(float), (vox.y - 1 + res.y) % res.y , vox.z);
	surf3Dread(&vals[3], surfIn, vox.x * sizeof(float), (vox.y + 1) % res.y, vox.z);
	surf3Dread(&vals[4], surfIn, vox.x * sizeof(float), vox.y, (vox.z - 1 + res.z) % res.z);
	surf3Dread(&vals[5], surfIn, vox.x * sizeof(float), vox.y, (vox.z + 1) % res.z);

	float valCenter;
	surf3Dread(&valCenter, surfIn, vox.x * sizeof(float), vox.y, vox.z);

	float newVal = valCenter;
	bool isInside = ((vals[0] > 0.0f) && (vals[1] > 0.0f) && (vals[2] > 0.0f) && 
					(vals[3] > 0.0f) && (vals[4] > 0.0f) && (vals[5] > 0.0f));
	
	if (!isInside) {
		newVal = 0.0f;
	}	

	surf3Dwrite(newVal, surfOut, vox.x * sizeof(float), vox.y, vox.z);
}

void launchErodeKernel(uint3 res, hipSurfaceObject_t surfIn, hipSurfaceObject_t surfOut){

	uint3 block = make_uint3(8, 8, 8);
	uint3 numBlocks = make_uint3(
		(res.x / block.x) + 1,
		(res.y / block.y) + 1,
		(res.z / block.z) + 1
	);

	kernelErode << <numBlocks, block >> > (res, surfIn, surfOut);
}



template <int blockSize, int apron>
__global__ void kernelHeat(uint3 res, hipSurfaceObject_t surfIn, hipSurfaceObject_t surfOut) {

	static_assert(apron * 2 < blockSize, "Apron must be less than blockSize / 2");
	const int N = blockSize;
	const int3 tid = make_int3(threadIdx.x, threadIdx.y, threadIdx.z);

	//Sliding window of blockdim - 2*apron size
	const int3 vox = make_int3(
		blockIdx.x * (blockDim.x - 2*apron), 
		blockIdx.y * (blockDim.y - 2*apron), 
		blockIdx.z * (blockDim.z - 2*apron)
	) + tid - make_int3(apron);

	//Toroidal boundaries
	const int3 voxToroid = make_int3((vox.x + res.x) % res.x, (vox.y + res.y) % res.y, (vox.z + res.z) % res.z);
	
	//Read whole block into shared memory
	__shared__ float ndx[N][N][N];
	surf3Dread(
		&ndx[tid.x][tid.y][tid.z],
		surfIn,
		voxToroid.x * sizeof(float), voxToroid.y, voxToroid.z
	);
	__syncthreads();
	
	//Skip apron voxels
	if (tid.x < apron || tid.x >= N - apron ||
		tid.y < apron || tid.y >= N - apron ||
		tid.z < apron || tid.z >= N - apron ) return;

	//Skip outside voxels
	if (vox.x >= res.x || vox.y >= res.y || vox.z >= res.z ||
		vox.x < 0 || vox.y < 0 || vox.z < 0)	return;

	/////////// Compute
		
	float oldVal = ndx[tid.x][tid.y][tid.z];
	/*
		Heaters
	*/
	int rad = 5;
	if (vox.x - res.x / 4 > res.x / 2 - rad && vox.y > res.y / 2 - rad && vox.z > res.z / 2 - rad &&
		vox.x - res.x / 4 < res.x / 2 + rad && vox.y < res.y / 2 + rad && vox.z < res.z / 2 + rad
		)
		oldVal += 1.5f;

	if (vox.x + res.x / 4 > res.x / 2 - rad && vox.y > res.y / 2 - rad && vox.z > res.z / 2 - rad &&
		vox.x + res.x / 4 < res.x / 2 + rad && vox.y < res.y / 2 + rad && vox.z < res.z / 2 + rad
		)
		oldVal += 1.5f;

	float dt = 0.1f;

	// New heat
	float newVal = oldVal + dt * (
		ndx[tid.x - 1][tid.y][tid.z] +
		ndx[tid.x + 1][tid.y][tid.z] +
		ndx[tid.x][tid.y - 1][tid.z] +
		ndx[tid.x][tid.y + 1][tid.z] +
		ndx[tid.x][tid.y][tid.z - 1] +
		ndx[tid.x][tid.y][tid.z + 1] -
		oldVal * 6.0f
		);

	surf3Dwrite(newVal, surfOut, vox.x * sizeof(float), vox.y, vox.z);	
	
}


void launchHeatKernel(uint3 res, hipSurfaceObject_t surfIn, hipSurfaceObject_t surfOut) {

	const int blockSize = 8;
	const int apron = 1;


	uint3 block = make_uint3(blockSize);	
	uint3 numBlocks = make_uint3(
		(res.x / (block.x - 2 * apron)) + 1,
		(res.y / (block.y - 2 * apron)) + 1,
		(res.z / (block.z - 2 * apron)) + 1
	);

	kernelHeat<blockSize,apron><< <numBlocks, block >> > (res, surfIn, surfOut);
}







__global__ void kernelBinarizeFloat(uint3 res, hipSurfaceObject_t surfInOut, float threshold) {

	VOLUME_VOX_GUARD(res);

	float val = 0.0f;
	surf3Dread(&val, surfInOut, vox.x * sizeof(float), vox.y, vox.z);	

	val = (val < threshold) ? 0.0f : 1.0f;	

	surf3Dwrite(val, surfInOut, vox.x * sizeof(float), vox.y, vox.z);
}

template <typename T>
__global__ void kernelBinarizeUnsigned(uint3 res, hipSurfaceObject_t surfInOut, T threshold) {

	VOLUME_VOX_GUARD(res);

	T val = 0;
	surf3Dread(&val, surfInOut, vox.x * sizeof(T), vox.y, vox.z);

	val = (val < threshold) ? T(0) : T(-1);

	surf3Dwrite(val, surfInOut, vox.x * sizeof(T), vox.y, vox.z);
}



void launchBinarizeKernel(uint3 res, hipSurfaceObject_t surfInOut, PrimitiveType type, float threshold) {

	uint3 block = make_uint3(8, 8, 8);
	uint3 numBlocks = make_uint3(
		(res.x / block.x) + 1,
		(res.y / block.y) + 1,
		(res.z / block.z) + 1
	);

	if (type == TYPE_FLOAT)
		kernelBinarizeFloat << <numBlocks, block >> > (res, surfInOut, threshold);
	else if (type == TYPE_UCHAR)
		kernelBinarizeUnsigned<uchar> << <numBlocks, block >> > (res, surfInOut, uchar(threshold * 255));
	else
		exit(-1);
}


template <int blockSize, int apron>
__global__ void kernelDiffuse(DiffuseParams params) {	
	static_assert(apron * 2 < blockSize, "Apron must be less than blockSize / 2");
	const uint3 res = params.res;
	const int N = blockSize;
	const int3 tid = make_int3(threadIdx.x, threadIdx.y, threadIdx.z);

	//Sliding window of blockdim - 2*apron size
	const int3 vox = make_int3(
		blockIdx.x * (blockDim.x - 2 * apron),
		blockIdx.y * (blockDim.y - 2 * apron),
		blockIdx.z * (blockDim.z - 2 * apron)
	) + tid - make_int3(apron);

	//Toroidal boundaries	

	//Read whole block into shared memory
	__shared__ float ndx[N][N][N];
	__shared__ float Ddx[N][N][N];

	
	//Priority x > y > z (instead of 27 boundary values, just use 6)	
	Dir dir = DIR_NONE;
	if (vox.x < 0) 		
		dir = X_NEG;	
	else if (vox.x >= res.x) 
		dir = X_POS;
	else if (vox.y < 0)
		dir = Y_NEG;
	else if (vox.y >= res.y)
		dir = Y_POS;
	else if (vox.z < 0)
		dir = Z_NEG;
	else if (vox.z >= res.z)
		dir = Z_POS;
	

	if (dir != DIR_NONE) {
		ndx[tid.x][tid.y][tid.z] = params.boundaryValues[dir];
		Ddx[tid.x][tid.y][tid.z] = BOUNDARY_ZERO_GRADIENT;
	}
	else {
		surf3Dread(
			&ndx[tid.x][tid.y][tid.z],
			params.concetrationIn,
			vox.x * sizeof(float), vox.y, vox.z
		);

		uchar maskVal;
		surf3Dread(
			&maskVal,
			params.mask,
			vox.x * sizeof(uchar), vox.y, vox.z
		);
		if (maskVal == 0)
			Ddx[tid.x][tid.y][tid.z] = params.zeroDiff;
		else
			Ddx[tid.x][tid.y][tid.z] = params.oneDiff;


	}	
	__syncthreads();

	//If zero grad boundary cond, copy value from neighbour (after sync!)
	if (ndx[tid.x][tid.y][tid.z] == BOUNDARY_ZERO_GRADIENT) {		
		int3 neighVec = -dirVec(dir);
		ndx[tid.x][tid.y][tid.z] = ndx[tid.x + neighVec.x][tid.y + neighVec.y][tid.z + neighVec.z];
	}

	if (Ddx[tid.x][tid.y][tid.z] == BOUNDARY_ZERO_GRADIENT) {
		int3 neighVec = -dirVec(dir);
		Ddx[tid.x][tid.y][tid.z] = Ddx[tid.x + neighVec.x][tid.y + neighVec.y][tid.z + neighVec.z];
	}
	//TODO: test what is faster -> double read from global memory, or copy within shared with extra threadsync

	__syncthreads();


	//Skip apron voxels
	if (tid.x < apron || tid.x >= N - apron ||
		tid.y < apron || tid.y >= N - apron ||
		tid.z < apron || tid.z >= N - apron) return;

	//Skip outside voxels
	if (vox.x >= res.x || vox.y >= res.y || vox.z >= res.z ||
		vox.x < 0 || vox.y < 0 || vox.z < 0)	return;

	//Load battery value
	
	uchar mask = 0;
	surf3Dread(&mask, params.mask, vox.x * sizeof(uchar), vox.y, vox.z);

	//Diffusion coeff
	float D = (mask == 0) ? params.zeroDiff : params.oneDiff;	



	
	///
	{
		float dx = params.voxelSize;

		const float D = Ddx[tid.x][tid.y][tid.z];
		const float3 D3 = make_float3(D);

		const float3 Dneg = lerp(
			D3,
			make_float3(Ddx[tid.x - 1][tid.y][tid.z], Ddx[tid.x][tid.y - 1][tid.z], Ddx[tid.x][tid.y][tid.z-1]),
			(dx * 0.5f)
		);

		const float3 Dpos = lerp(
			D3,
			make_float3(Ddx[tid.x + 1][tid.y][tid.z], Ddx[tid.x][tid.y + 1][tid.z], Ddx[tid.x][tid.y][tid.z + 1]),			
			(dx * 0.5f)
		);	


		const float3 C = make_float3(ndx[tid.x][tid.y][tid.z]);

		const float3 Cneg = lerp(
			C,
			make_float3(ndx[tid.x - 1][tid.y][tid.z], ndx[tid.x][tid.y - 1][tid.z],	ndx[tid.x][tid.y][tid.z - 1]),
			dx
		);

		const float3 Cpos = lerp(
			C,
			make_float3(ndx[tid.x + 1][tid.y][tid.z], ndx[tid.x][tid.y + 1][tid.z], ndx[tid.x][tid.y][tid.z + 1]),
			dx
		);
		
		float dt = dx*dx * (1.0f / (2.0f * min(params.zeroDiff, params.oneDiff)));


		//https://math.stackexchange.com/questions/1949795/explicit-finite-difference-scheme-for-nonlinear-diffusion
		//float3 dc = (dt / (dx*dx)) * (Dpos * (Cpos - C) - Dneg * (C - Cneg));

		float3 dc = Dneg * Cneg + Dpos * Cpos - C * (Dneg + Dpos);

		//float3 dc = D * (Cpos - 2* C + Cneg) + (Dneg - Dpos) * ()

		//if (vox.x == 2 && vox.y == 10 && vox.z == 10) {
			//printf("dt: %f\n", dt);
		//}
			//printf("c: %f, D: %.9f dc: %f %f %f, Dneg: %f %f %f\n",C.x, D, dc.x, dc.y, dc.z, Dneg.x, Dneg.y, Dneg.z);

		float DX = 1.0f / res.x;

		float newVal = C.x + (dc.x + dc.y + dc.z);
		

		surf3Dwrite(newVal, params.concetrationOut, vox.x * sizeof(float), vox.y, vox.z);


		return;

		//float3 dD2 = make_float3(
		//	Ddx[tid.x - 1][tid.y][tid.z] + 2.0f * oldVal + Ddx[tid.x + 1][tid.y][tid.z],
		//	Ddx[tid.x][tid.y - 1][tid.z] + 2.0f * oldVal + Ddx[tid.x][tid.y + 1][tid.z],
		//	Ddx[tid.x][tid.y][tid.z - 1] + 2.0f * oldVal + Ddx[tid.x][tid.y][tid.z + 1]
		//);

		//float3 dc2 = make_float3(
		//	ndx[tid.x - 1][tid.y][tid.z] + 2.0f * oldVal + ndx[tid.x + 1][tid.y][tid.z],
		//	ndx[tid.x][tid.y - 1][tid.z] + 2.0f * oldVal + ndx[tid.x][tid.y + 1][tid.z],
		//	ndx[tid.x][tid.y][tid.z - 1] + 2.0f * oldVal + ndx[tid.x][tid.y][tid.z + 1]
		//);


		

	}

	/////////// Compute

	float oldVal = ndx[tid.x][tid.y][tid.z];

	//http://janroman.dhis.org/finance/Numerical%20Methods/adi.pdf
	//float dt = 0.1f;
	int minDim = min(res.x, min(res.y, res.z));
	//float dt = 1.0f / (6.0f * minDim * D);
	//float3 dX = make_float3(1.0f / (res.x), 1.0f / (res.y), 1.0f / (res.z));
	float3 dX = make_float3(0.37e-6f);// , 1.0f / (res.y), 1.0f / (res.z));
	float3 dX2 = make_float3(dX.x*dX.x, dX.y*dX.y, dX.z*dX.z);


	float minD = max(params.zeroDiff, params.oneDiff);

	float dt = 1.0f / (2.0f * minD * (1.0f / dX2.x +  1.0f / dX2.y + 1.0f / dX2.z));
	
	//dt *= 1.0f / 10.0f;
	float3 v = D * make_float3(dt / dX2.x , dt / dX2.y, dt / dX2.z);

	//D(dt / 1 + dt / 1 + dt / 1) <= 1/2 >>> dt <= 1/6*D
	//3 * dt / (1/64)^2 <= 1/2 >>> dt <= 1/6 * 1/4096 >>> dt <= 1 / 24576*D 
	//D * dt * (1 / resx^2 + 1 / resy^2 + 1 / resz^2) <= 1/2
		//>>> dt <= 1/2 * 1/D * 1/ sum(dX.x^2)


//	if (vox.x < 2 && vox.y == 10 && vox.z == 10)
	//	printf("x %d, dt %.9f, vsum %.9f, val: %f dx: %f\n", vox.x ,dt , v.x+v.y+v.z, oldVal, oldVal - ndx[tid.x - 1][tid.y][tid.z]);

	
	float3 d2 = make_float3(
		ndx[tid.x - 1][tid.y][tid.z] + 2.0f * oldVal + ndx[tid.x + 1][tid.y][tid.z],
		ndx[tid.x][tid.y - 1][tid.z] + 2.0f * oldVal + ndx[tid.x][tid.y + 1][tid.z],
		ndx[tid.x][tid.y][tid.z - 1] + 2.0f * oldVal + ndx[tid.x][tid.y][tid.z + 1]
	);

	float newVal = oldVal +  (v.x * d2.x + v.y * d2.y + v.z * d2.z);
	
	//newVal = oldVal + v.x;
		

	surf3Dwrite(newVal, params.concetrationOut, vox.x * sizeof(float), vox.y, vox.z);

}

void launchDiffuseKernel(DiffuseParams params) {

	const int blockSize = 8;
	const int apron = 1;

	uint3 res = params.res;

	uint3 block = make_uint3(blockSize);
	uint3 numBlocks = make_uint3(
		(res.x / (block.x - 2 * apron)) + 1,
		(res.y / (block.y - 2 * apron)) + 1,
		(res.z / (block.z - 2 * apron)) + 1
	);

	kernelDiffuse<blockSize, apron> << <numBlocks, block >> > (params);
}







__global__ void kernelSubtract(uint3 res, hipSurfaceObject_t A, hipSurfaceObject_t B) {

	VOLUME_VOX_GUARD(res);

	float Aval, Bval;
	surf3Dread(&Aval, A, vox.x * sizeof(float), vox.y, vox.z);
	surf3Dread(&Bval, B, vox.x * sizeof(float), vox.y, vox.z);

	float newVal = Bval - Aval;
	
	surf3Dwrite(newVal, A, vox.x * sizeof(float), vox.y, vox.z);
}

void launchSubtractKernel(uint3 res, hipSurfaceObject_t A, hipSurfaceObject_t B) {
	uint3 block = make_uint3(8, 8, 8);
	uint3 numBlocks = make_uint3(
		(res.x / block.x) + 1,
		(res.y / block.y) + 1,
		(res.z / block.z) + 1
	);

	kernelSubtract << <numBlocks, block >> > (res, A, B);

}



//template <typename T, unsigned int blockSize>
//__global__ void reduce(T *g_idata, T *g_odata, unsigned int n)
//{
//	extern __shared__ int sdata[];
//	unsigned int tid = threadIdx.x;
//	unsigned int i = blockIdx.x*(blockSize * 2) + tid;
//	unsigned int gridSize = blockSize * 2 * gridDim.x;
//	sdata[tid] = 0;
//
//	while (i < n) { sdata[tid] += g_idata[i] + g_idata[i + blockSize]; i += gridSize; }
//	__syncthreads();
//
//	if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
//	if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
//	if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
//	if (tid < 32) {
//		if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
//		if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
//		if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
//		if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
//		if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
//		if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
//	}
//	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
//}

__host__ __device__ uint3 ind2sub(uint3 res, uint i) {
	uint x = i % res.x;
	uint tmp = ((i - x) / res.x);
	uint y = tmp % res.y;
	uint z = (tmp - y) / res.y;
	return make_uint3(
		x,y,z
	);
}


template <typename T, unsigned int blockSize, bool toSurface>
__global__ void reduce3D(uint3 res, hipSurfaceObject_t data, T * finalData, unsigned int n, uint3 offset)
{
	extern __shared__ T sdata[];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockSize * 2) + tid;
	unsigned int gridSize = blockSize * 2 * gridDim.x;
	sdata[tid] = 0;

	while (i < n) { 
		uint3 voxi = ind2sub(res, i);
		uint3 voxip = ind2sub(res, i + blockSize); 

		T vali, valip = 0;
		surf3Dread(&vali, data, voxi.x * sizeof(T), voxi.y, voxi.z);
		if(voxip.x < res.x && voxip.y < res.y && voxip.z < res.z)
			surf3Dread(&valip, data, voxip.x * sizeof(T), voxip.y, voxip.z);	
				
	
		sdata[tid] += (vali + valip);

		i += gridSize; 
	}
	__syncthreads();

	if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
	if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
	if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
	if (tid < 32) {
		if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
		if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
		if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
		if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
		if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
		if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
	}
	if (tid == 0) {
		unsigned int o = blockIdx.x;	
		//Either copy to surface
		if (toSurface) {						
			uint3 voxo = ind2sub(res,o);
			surf3Dwrite(sdata[0], data, voxo.x * sizeof(T), voxo.y, voxo.z);				
		}
		//Or final 1D array
		else {
			finalData[o] = sdata[0];						
		}	
	}
	
}



float launchReduceSumKernel(uint3 res, hipSurfaceObject_t surf) {
		

	const uint finalSizeMax = 512;	
	const uint blockSize = 512;
	const uint3 block = make_uint3(blockSize,1,1);	
	uint n = res.x * res.y * res.z;

	//uint finalSizeMax = ((res.x * res.y * res.z) / blockSize) / 2;


	float * deviceResult = nullptr;
	hipMalloc(&deviceResult, finalSizeMax * sizeof(float));
	hipMemset(deviceResult, 0, finalSizeMax * sizeof(float));
	

	while (n > finalSizeMax) {
		uint3 numBlocks = make_uint3(
			(n / block.x) / 2 , 1, 1
		);		

		//If not final stage of reduction -> reduce into  surface
		if (numBlocks.x > finalSizeMax) {
			reduce3D<float, blockSize, true>
				<<<numBlocks, block, blockSize * sizeof(float)>>> (
					res, surf, nullptr, n, make_uint3(0)
					);
		}
		else {
			reduce3D<float, blockSize, false>
				<<<numBlocks, block, blockSize * sizeof(float)>>> (
					res, surf, deviceResult, n, make_uint3(0)
					);
		
		}

		//New N
		n = numBlocks.x;
	}


	float * hostResult = new float[finalSizeMax];
	hipMemcpy(hostResult, deviceResult, finalSizeMax * sizeof(float), hipMemcpyDeviceToHost);


	float result = 0.0f;
	for (auto i = 0; i < finalSizeMax; i++) {
		result += hostResult[i];
	}

	hipFree(deviceResult);
	delete[] hostResult;


	return result;

}




//////////////////////////

template <Dir dir>
__global__ void sliceReduce(uint3 res, hipSurfaceObject_t surf, float *output) {
		
	const uint3 tid = make_uint3(threadIdx.x, threadIdx.y, threadIdx.z);
	const uint tidLin = tid.x + tid.y * blockDim.x; 

	const uint3 vox = make_uint3(
		blockIdx.x * blockDim.x, //in slice x
		blockIdx.y * blockDim.y, //in slice y
		blockIdx.z * blockDim.z  //slice
	) + tid;

	{
		/*uint blockIndex = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
		if(blockIndex == 0 || blockIndex == 127)
			printf("block: %d %d %d - > %d, vox: %d %d %d | blockDim: %d %d %d\n", 
				blockIdx.x, blockIdx.y, blockIdx.z,
				blockIndex, 
				vox.x, vox.y, vox.z,
				blockDim.x, blockDim.y, blockDim.z
			);*/
	}

	extern __shared__ float s[];

	uint3 texVox;
	if (dir == Z_POS || dir == Z_NEG)
		texVox = vox;
	else if (dir == Y_POS || dir == Y_POS)
		texVox = make_uint3(vox.x, vox.z, vox.y);
	else 
		texVox = make_uint3(vox.y, vox.z, vox.x);

	s[tidLin] = 0.0f;


	bool valid = true;
	if (texVox.x >= res.x || texVox.y > res.y || texVox.z >= res.z) {
		valid = false;
	}

	if(valid)
		surf3Dread(&s[tidLin], surf, texVox.x * sizeof(float), texVox.y, texVox.z);

	__syncthreads();

	
	//For now let tid==0 do all the work
	if (tidLin == 0) {
		float blockResult = 0.0f;
		uint perBlock = blockDim.x * blockDim.y;
		for (uint i = 0; i < perBlock; i++) {
			blockResult += s[i];
		}

		uint blockIndex = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
		output[blockIndex] = blockResult;

	}

	
}


float launchReduceSumSlice(uint3 res, hipSurfaceObject_t surf, Dir dir, void * output){
	



	const uint3 blockSize = make_uint3(16,16,1);
	const uint dirIndex = getDirIndex(dir);

	uint *resArr = (uint*)&res;
	uint3 resRotated = make_uint3(
		resArr[(dirIndex + 1) % 3], 
		resArr[(dirIndex + 2) % 3], 
		resArr[dirIndex]
	);

	//uint3 resRotated = make_uint3(864, 864, 289);

	const uint3 grid = roundDiv(resRotated, blockSize);

	const uint size = grid.x * grid.y * grid.z;

	float * deviceResult = nullptr;
	hipMalloc(&deviceResult, size * sizeof(float));
	hipMemset(deviceResult, 0, size * sizeof(float));

	if(dirIndex == 0)
		sliceReduce<X_POS> << <grid, blockSize, blockSize.x * blockSize.y * sizeof(float) >> >
			(res, surf, deviceResult);
	else if(dirIndex == 1)
		sliceReduce<Y_POS> << <grid, blockSize, blockSize.x * blockSize.y * sizeof(float) >> >
		(res, surf, deviceResult);
	else if (dirIndex == 2)
		sliceReduce<Z_POS> << <grid, blockSize, blockSize.x * blockSize.y * sizeof(float) >> >
		(res, surf, deviceResult);

		 

	float * hostResult = new float[size];
	hipMemcpy(hostResult, deviceResult, size * sizeof(float), hipMemcpyDeviceToHost);


	for (auto sliceID = 0; sliceID < grid.z; sliceID++) {

		((float *)output)[sliceID] = 0.0f;
		for (auto k = 0; k < grid.x * grid.y; k++) {
			((float *)output)[sliceID] += hostResult[grid.x * grid.y * sliceID + k];
		}

	}

	hipFree(deviceResult);
	delete[] hostResult;





	//uint3 grid = make_uint3(resRotated.x / blockSize.x, roundDiv(resRotated.y, blockSize.y), resRotated.z / blockSize.z);	
	/*uint finalSizeMax = 1;
	while (sliceNum > finalSizeMax) 
		finalSizeMax *= 2;
	
	

	float * deviceResult = nullptr;
	hipMalloc(&deviceResult, finalSizeMax * sizeof(float));
	hipMemset(deviceResult, 0, finalSizeMax * sizeof(float));




	hipFree(deviceResult);*/

	return 0;

}








/*
	Surface & buffer reduction, templated	
*/


template <typename T>
__device__ void opSum(volatile T & a, T b) {
	a += b;
}

template <typename T>
__device__ void opMin(volatile T & a, T b) {
	if (b < a) a = b;
}
template <typename T>
__device__ void opMax(volatile T & a, T  b) {
	if (b > a) a = b;
}



template <typename T>
using ReduceOp = void(*)(
	volatile T & a, T b
	);

template <typename T>
__device__ void opSquare(T & a) {
	a *= a;
}

template <typename T>
__device__ void opIdentity(T & a) {
	//nothing
}

template <typename T>
using PreReduceOp = void(*)(
	T & a
	);

template <typename T, unsigned int blockSize, ReduceOp<T> _op, PreReduceOp<T> _preOp = opIdentity<T>>
__global__ void reduce3DSurfaceToBuffer(uint3 res, hipSurfaceObject_t surf, T * reducedData, size_t n)
{
	extern __shared__ __align__(sizeof(T)) volatile unsigned char my_smem[];
	volatile T *sdata = reinterpret_cast<volatile T *>(my_smem);

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockSize * 2) + tid;
	unsigned int gridSize = blockSize * 2 * gridDim.x;
	sdata[tid] = T(0);

	while (i < n) {
		const uint3 voxi = ind2sub(res, i);
		const uint3 voxip = ind2sub(res, i + blockSize);

		if (voxi.x < res.x && voxi.y < res.y && voxi.z < res.z) {
			
			/*if (threadIdx.x == 32 && threadIdx.y == 0 && threadIdx.z == 0) {
				if (blockIdx.x == 16 && blockIdx.y == 0 && blockIdx.z == 0) {
					printf("%d: %d %d %d ... %d %d %d\n",i, res.x, res.y, res.z , voxi.x, voxi.y, voxi.z);
				}
			}*/

			T vali = T(0);
			vali = read<T>(surf, voxi);
			_preOp(vali);
			_op(sdata[tid], vali);
		}

		if (i + blockSize < n && voxip.x < res.x && voxip.y < res.y && voxip.z < res.z) {
			T valip = T(0);
			valip = read<T>(surf, voxip);
			_preOp(valip);
			_op(sdata[tid], valip);
		}		

		i += gridSize;
	}
	__syncthreads();

	if (blockSize >= 512) { if (tid < 256) { _op(sdata[tid], sdata[tid + 256]); } __syncthreads(); }
	if (blockSize >= 256) { if (tid < 128) { _op(sdata[tid], sdata[tid + 128]); } __syncthreads(); }
	if (blockSize >= 128) { if (tid < 64) { _op(sdata[tid], sdata[tid + 64]); } __syncthreads(); }
	if (tid < 32) {
		if (blockSize >= 64) _op(sdata[tid], sdata[tid + 32]);
		if (blockSize >= 32) _op(sdata[tid], sdata[tid + 16]);
		if (blockSize >= 16) _op(sdata[tid], sdata[tid + 8]);
		if (blockSize >= 8) _op(sdata[tid], sdata[tid + 4]);
		if (blockSize >= 4) _op(sdata[tid], sdata[tid + 2]);
		if (blockSize >= 2) _op(sdata[tid], sdata[tid + 1]);
	}

	if (tid == 0) {		
		reducedData[blockIdx.x] = sdata[0];
	}

}


template <typename T, unsigned int blockSize, ReduceOp<T> _op>
__global__ void reduceBuffer(T * buffer, size_t n)
{
	extern __shared__ __align__(sizeof(T)) volatile unsigned char my_smem[];
	volatile T *sdata = reinterpret_cast<volatile T *>(my_smem);
	
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockSize * 2) + tid;
	unsigned int gridSize = blockSize * 2 * gridDim.x;
	sdata[tid] = 0;

	while (i < n) {
		_op(sdata[tid], buffer[i]);

		if(i + blockSize < n)
			_op(sdata[tid], buffer[i + blockSize]);

		i += gridSize;
	}
	__syncthreads();

	if (blockSize >= 512) { if (tid < 256) { _op(sdata[tid], sdata[tid + 256]); } __syncthreads(); }
	if (blockSize >= 256) { if (tid < 128) { _op(sdata[tid], sdata[tid + 128]); } __syncthreads(); }
	if (blockSize >= 128) { if (tid < 64) { _op(sdata[tid], sdata[tid + 64]); } __syncthreads(); }
	if (tid < 32) {
		if (blockSize >= 64) _op(sdata[tid], sdata[tid + 32]);
		if (blockSize >= 32) _op(sdata[tid], sdata[tid + 16]);
		if (blockSize >= 16) _op(sdata[tid], sdata[tid + 8]);
		if (blockSize >= 8) _op(sdata[tid], sdata[tid + 4]);
		if (blockSize >= 4) _op(sdata[tid], sdata[tid + 2]);
		if (blockSize >= 2) _op(sdata[tid], sdata[tid + 1]);
	}

	if (tid == 0) {
		buffer[blockIdx.x] = sdata[0];
	}

}



//AuxBuffer -> surf total n / 512
void launchReduceKernel(	
	PrimitiveType type, 
	ReduceOpType opType,
	uint3 res, 
	hipSurfaceObject_t surf, 
	void * auxBufferGPU,
	void * auxBufferCPU,
	void * result	
) {

	const uint blockSize = VOLUME_REDUCTION_BLOCKSIZE;
	const uint sharedSize = primitiveSizeof(type) * blockSize;
	
	const uint3 block = make_uint3(blockSize, 1, 1);
	const uint finalSizeMax = VOLUME_REDUCTION_BLOCKSIZE;
	const size_t initialN = res.x * res.y * res.z;
	
	size_t n = initialN;
		
	/*
		Reduce from surface to auxiliar buffer
	*/
	{		
		uint3 numBlocks = make_uint3(
			(n / block.x) / 2, 1, 1
		);
		if (numBlocks.x == 0)
			numBlocks.x = 1;

		if (type == TYPE_FLOAT) {
			if (opType == REDUCE_OP_SQUARESUM)
				reduce3DSurfaceToBuffer<float, blockSize, opSum, opSquare> << <numBlocks, block, sharedSize>> > (
					res, surf, (float*)auxBufferGPU, n
					);						
		}
		else if (type == TYPE_DOUBLE) {
			if (opType == REDUCE_OP_SQUARESUM) {
				reduce3DSurfaceToBuffer<double, blockSize, opSum, opSquare> << <numBlocks, block, sharedSize >> > (
					res, surf, (double*)auxBufferGPU, n
					);
			}			

		}

		n = numBlocks.x;
	}


	/*
		Further reduce in buffer
	*/
	while (n > finalSizeMax) {
		const uint blockSize = VOLUME_REDUCTION_BLOCKSIZE;
		const uint3 block = make_uint3(blockSize, 1, 1);
		uint3 numBlocks = make_uint3(
				(n / block.x) / 2, 1, 1
			);

		if (type == TYPE_FLOAT) {
			if (opType == REDUCE_OP_SQUARESUM)
				reduceBuffer<float, blockSize, opSum><<<numBlocks,block, sharedSize>>>((float*)auxBufferGPU, n);				
		}
		if (type == TYPE_DOUBLE) {
			if (opType == REDUCE_OP_SQUARESUM)
				reduceBuffer<double, blockSize, opSum> << <numBlocks, block, sharedSize >> >((double*)auxBufferGPU, n);
		}


		n = numBlocks.x;
	}
	

	
	hipMemcpy(auxBufferCPU, auxBufferGPU, primitiveSizeof(type) * n, hipMemcpyDeviceToHost);

	/*
		Sum last array on CPU
	*/
	if (type == TYPE_FLOAT) {	
		*((float*)result) = 0.0f;
		
		for (auto i = 0; i < n; i++) {
			//printf("%f\n", ((float*)auxBufferCPU)[i]);			
			*((float*)result) += ((float*)auxBufferCPU)[i];
		}
	}
	else if (type == TYPE_DOUBLE) {	
		*((double*)result) = 0.0;
		for (auto i = 0; i < n; i++) {
			*((double*)result) += ((double*)auxBufferCPU)[i];
		}		
	}

	
	char b;
	b = 0;

	return;


	//float * deviceResult = nullptr;
	//hipMalloc(&deviceResult, finalSizeMax * sizeof(float));
	//hipMemset(deviceResult, 0, finalSizeMax * sizeof(float));


	//while (n > finalSizeMax) {
	//	uint3 numBlocks = make_uint3(
	//		(n / block.x) / 2, 1, 1
	//	);

	//	//If not final stage of reduction -> reduce into  surface
	//	if (numBlocks.x > finalSizeMax) {
	//		reduce3D<float, blockSize, true>
	//			<< <numBlocks, block, blockSize * sizeof(float) >> > (
	//				res, surf, nullptr, n, make_uint3(0)
	//				);
	//	}
	//	else {
	//		reduce3D<float, blockSize, false>
	//			<< <numBlocks, block, blockSize * sizeof(float) >> > (
	//				res, surf, deviceResult, n, make_uint3(0)
	//				);

	//	}

	//	//New N
	//	n = numBlocks.x;
	//}


	//float * hostResult = new float[finalSizeMax];
	//hipMemcpy(hostResult, deviceResult, finalSizeMax * sizeof(float), hipMemcpyDeviceToHost);


	//float result = 0.0f;
	//for (auto i = 0; i < finalSizeMax; i++) {
	//	result += hostResult[i];
	//}

	//hipFree(deviceResult);
	//delete[] hostResult;


	//return result;

}

